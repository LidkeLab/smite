#include "hip/hip_runtime.h"
/*!
 * \file GPUgaussMLEv2.cu
 * \author Keith Lidke
 * \date January 10, 2010
 * \brief This file contains all of the Cuda kernels.  The helper functions
 * are defined in GPUgaussLib.cuh
 */
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "mex.h"
#include "hip/hip_runtime.h"
#include "definitions.h"
#include "MatInvLib.h"
#include "GPUgaussLib.cuh"
#include "GPUgaussMLEv2.h"

//*******************************************************************************************
//theta is: {x,y,N,bg}
__global__ void kernel_MLEFit_noshared(const float *d_data, const float PSFSigma, const int sz, const int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,const int Nfits){
	/*! 
	 * \brief basic MLE fitting kernel.  No additional parameters are computed.
	 * \param d_data array of subregions to fit copied to GPU
	 * \param PSFSigma sigma of the point spread function
	 * \param sz nxn size of the subregion to fit
	 * \param iterations number of iterations for solution to converge
	 * \param d_Parameters array of fitting parameters to return for each subregion
	 * \param d_CRLBs array of Cramer-Rao lower bound estimates to return for each subregion
	 * \param d_LogLikelihood array of loglikelihood estimates to return for each subregion
	 * \param Nfits number of subregions to fit
	 */
    //__shared__ float s_data[MEM];
    float M[NV_P*NV_P], Diag[NV_P], Minv[NV_P*NV_P];
    const int tx = threadIdx.x;
    const int bx = blockIdx.x;
    const int BlockSize = blockDim.x;
    int ii, jj, kk, ll;
    float model, cf, df, data;
    float Div;
    float PSFy, PSFx;
    int NV=NV_P;
    float dudt[NV_P];
    float d2udt2[NV_P];
    float NR_Numerator[NV_P], NR_Denominator[NV_P];
    float theta[NV_P];
    float maxjump[NV_P]={1e0f, 1e0f, 1e2f, 2e0f};
    float gamma[NV_P]={1.0f, 1.0f, 0.5f, 1.0f};
    float Nmax;

    //Prevent read/write past end of array
    if ((bx*BlockSize+tx)>=Nfits) return;
    
	memset(M,0,NV_P*NV_P*sizeof(float));
	memset(Minv,0,NV_P*NV_P*sizeof(float));
    //load data
    const float *s_data = d_data+(sz*sz*bx*BlockSize+sz*sz*tx);
    //initial values
    kernel_CenterofMass2D(sz, s_data, &theta[0], &theta[1]);
    kernel_GaussFMaxMin2D(sz, PSFSigma, s_data, &Nmax, &theta[3]);
    theta[2]=max(0.0f, (Nmax-theta[3])*2*pi*PSFSigma*PSFSigma);
    
    for (kk=0;kk<iterations;kk++) {//main iterative loop
        
        //initialize
		memset(NR_Numerator,0,NV_P*sizeof(float));
		memset(NR_Denominator,0,NV_P*sizeof(float));

        for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
            PSFx=kernel_IntGauss1D(ii, theta[0], PSFSigma);
            PSFy=kernel_IntGauss1D(jj, theta[1], PSFSigma);
            
            model=theta[3]+theta[2]*PSFx*PSFy;
            data=s_data[sz*jj+ii];
            
            //calculating derivatives
            kernel_DerivativeIntGauss1D(ii, theta[0], PSFSigma, theta[2], PSFy, &dudt[0], &d2udt2[0]);
            kernel_DerivativeIntGauss1D(jj, theta[1], PSFSigma, theta[2], PSFx, &dudt[1], &d2udt2[1]);
            dudt[2] = PSFx*PSFy;
            d2udt2[2] = 0.0f;
            dudt[3] = 1.0f;
            d2udt2[3] = 0.0f;
            
            cf=0.0f;
            df=0.0f;
            if (model>10e-3f) cf=data/model-1;
            if (model>10e-3f) df=data/pow(model, 2);
            cf=min(cf, 10e4f);
            df=min(df, 10e4f);
            
            for (ll=0;ll<NV;ll++){
                NR_Numerator[ll]+=dudt[ll]*cf;
                NR_Denominator[ll]+=d2udt2[ll]*cf-pow(dudt[ll], 2)*df;
            }
        }
        
        // The update
        if (kk<2)
            for (ll=0;ll<NV;ll++)
                theta[ll]-=gamma[ll]*min(max(NR_Numerator[ll]/NR_Denominator[ll], -maxjump[ll]), maxjump[ll]);
        else
            for (ll=0;ll<NV;ll++)
                theta[ll]-=min(max(NR_Numerator[ll]/NR_Denominator[ll], -maxjump[ll]), maxjump[ll]);
        
        // Any other constraints
        theta[2]=max(theta[2], 1.0f);
        theta[3]=max(theta[3], 0.01f);
        
    }
    
    // Calculating the CRLB and LogLikelihood
    Div=0.0;
    for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
        PSFx=kernel_IntGauss1D(ii, theta[0], PSFSigma);
        PSFy=kernel_IntGauss1D(jj, theta[1], PSFSigma);
        
        model=theta[3]+theta[2]*PSFx*PSFy;
        data=s_data[sz*jj+ii];
        
        //calculating derivatives
        kernel_DerivativeIntGauss1D(ii, theta[0], PSFSigma, theta[2], PSFy, &dudt[0], NULL);
        kernel_DerivativeIntGauss1D(jj, theta[1], PSFSigma, theta[2], PSFx, &dudt[1], NULL);
        dudt[2] = PSFx*PSFy;
        dudt[3] = 1.0f;
        
        //Building the Fisher Information Matrix
        for (kk=0;kk<NV;kk++)for (ll=kk;ll<NV;ll++){
            M[kk*NV+ll]+= dudt[ll]*dudt[kk]/model;
            M[ll*NV+kk]=M[kk*NV+ll];
        }
        
        //LogLikelyhood
        if (model>0)
            if (data>0)Div+=data*log(model)-model-data*log(data)+data;
            else
                Div+=-model;
    }
    
    // Matrix inverse (CRLB=F^-1) and output assigments
    kernel_MatInvN(M, Minv, Diag, NV);
    
    //write to global arrays
    for (kk=0;kk<NV;kk++) d_Parameters[Nfits*kk+BlockSize*bx+tx]=theta[kk];
    for (kk=0;kk<NV;kk++) d_CRLBs[Nfits*kk+BlockSize*bx+tx]=Diag[kk];
    d_LogLikelihood[BlockSize*bx+tx] = Div;
    
    return;
}

//*******************************************************************************************
//theta is: {x,y,N,bg}
__global__ void kernel_MLEFit(const float *d_data, const float PSFSigma, const int sz, const int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,const int Nfits){
	/*! 
	 * \brief basic MLE fitting kernel.  No additional parameters are computed.
	 * \param d_data array of subregions to fit copied to GPU
	 * \param PSFSigma sigma of the point spread function
	 * \param sz nxn size of the subregion to fit
	 * \param iterations number of iterations for solution to converge
	 * \param d_Parameters array of fitting parameters to return for each subregion
	 * \param d_CRLBs array of Cramer-Rao lower bound estimates to return for each subregion
	 * \param d_LogLikelihood array of loglikelihood estimates to return for each subregion
	 * \param Nfits number of subregions to fit
	 */
    __shared__ float s_data[MEM];
    float M[NV_P*NV_P], Diag[NV_P], Minv[NV_P*NV_P];
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int BlockSize = blockDim.x;
    int ii, jj, kk, ll;
    float model, cf, df, data;
    float Div;
    float PSFy, PSFx;
    int NV=NV_P;
    float dudt[NV_P];
    float d2udt2[NV_P];
    float NR_Numerator[NV_P], NR_Denominator[NV_P];
    float theta[NV_P];
    float maxjump[NV_P]={1e0f, 1e0f, 1e2f, 2e0f};
    float gamma[NV_P]={1.0f, 1.0f, 0.5f, 1.0f};
    float Nmax;

    //Prevent read/write past end of array
    if ((bx*BlockSize+tx)>=Nfits) return;
    
	memset(M,0,NV_P*NV_P*sizeof(float));
	memset(Minv,0,NV_P*NV_P*sizeof(float));
    //load data
	memcpy(s_data+(sz*sz*tx), d_data+(sz*sz*bx*BlockSize+sz*sz*tx),sizeof(float)*sz*sz);
    
    //initial values
    kernel_CenterofMass2D(sz, &s_data[sz*sz*tx], &theta[0], &theta[1]);
    kernel_GaussFMaxMin2D(sz, PSFSigma, &s_data[sz*sz*tx], &Nmax, &theta[3]);
    theta[2]=max(0.0f, (Nmax-theta[3])*2*pi*PSFSigma*PSFSigma);
    
    for (kk=0;kk<iterations;kk++) {//main iterative loop
        
        //initialize
		memset(NR_Numerator,0,NV_P*sizeof(float));
		memset(NR_Denominator,0,NV_P*sizeof(float));

        for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
            PSFx=kernel_IntGauss1D(ii, theta[0], PSFSigma);
            PSFy=kernel_IntGauss1D(jj, theta[1], PSFSigma);
            
            model=theta[3]+theta[2]*PSFx*PSFy;
            data=s_data[sz*sz*tx+sz*jj+ii];
            
            //calculating derivatives
            kernel_DerivativeIntGauss1D(ii, theta[0], PSFSigma, theta[2], PSFy, &dudt[0], &d2udt2[0]);
            kernel_DerivativeIntGauss1D(jj, theta[1], PSFSigma, theta[2], PSFx, &dudt[1], &d2udt2[1]);
            dudt[2] = PSFx*PSFy;
            d2udt2[2] = 0.0f;
            dudt[3] = 1.0f;
            d2udt2[3] = 0.0f;
            
            cf=0.0f;
            df=0.0f;
            if (model>10e-3f) cf=data/model-1;
            if (model>10e-3f) df=data/pow(model, 2);
            cf=min(cf, 10e4f);
            df=min(df, 10e4f);
            
            for (ll=0;ll<NV;ll++){
                NR_Numerator[ll]+=dudt[ll]*cf;
                NR_Denominator[ll]+=d2udt2[ll]*cf-pow(dudt[ll], 2)*df;
            }
        }
        
        // The update
        if (kk<2)
            for (ll=0;ll<NV;ll++)
                theta[ll]-=gamma[ll]*min(max(NR_Numerator[ll]/NR_Denominator[ll], -maxjump[ll]), maxjump[ll]);
        else
            for (ll=0;ll<NV;ll++)
                theta[ll]-=min(max(NR_Numerator[ll]/NR_Denominator[ll], -maxjump[ll]), maxjump[ll]);
        
        // Any other constraints
        theta[2]=max(theta[2], 1.0f);
        theta[3]=max(theta[3], 0.01f);
        
    }
    
    // Calculating the CRLB and LogLikelihood
    Div=0.0;
    for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
        PSFx=kernel_IntGauss1D(ii, theta[0], PSFSigma);
        PSFy=kernel_IntGauss1D(jj, theta[1], PSFSigma);
        
        model=theta[3]+theta[2]*PSFx*PSFy;
        data=s_data[sz*sz*tx+sz*jj+ii];
        
        //calculating derivatives
        kernel_DerivativeIntGauss1D(ii, theta[0], PSFSigma, theta[2], PSFy, &dudt[0], NULL);
        kernel_DerivativeIntGauss1D(jj, theta[1], PSFSigma, theta[2], PSFx, &dudt[1], NULL);
        dudt[2] = PSFx*PSFy;
        dudt[3] = 1.0f;
        
        //Building the Fisher Information Matrix
        for (kk=0;kk<NV;kk++)for (ll=kk;ll<NV;ll++){
            M[kk*NV+ll]+= dudt[ll]*dudt[kk]/model;
            M[ll*NV+kk]=M[kk*NV+ll];
        }
        
        //LogLikelyhood
        if (model>0)
            if (data>0)Div+=data*log(model)-model-data*log(data)+data;
            else
                Div+=-model;
    }
    
    // Matrix inverse (CRLB=F^-1) and output assigments
    kernel_MatInvN(M, Minv, Diag, NV);
    
    //write to global arrays
    for (kk=0;kk<NV;kk++) d_Parameters[Nfits*kk+BlockSize*bx+tx]=theta[kk];
    for (kk=0;kk<NV;kk++) d_CRLBs[Nfits*kk+BlockSize*bx+tx]=Diag[kk];
    d_LogLikelihood[BlockSize*bx+tx] = Div;
    
    return;
}

//*******************************************************************************************
__global__ void kernel_MLEFit_sigma(const float *d_data, const float PSFSigma, const int sz, const int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,const int Nfits){
	/*! 
	 * \brief basic MLE fitting kernel.  No additional parameters are computed.
	 * \param d_data array of subregions to fit copied to GPU
	 * \param PSFSigma sigma of the point spread function
	 * \param sz nxn size of the subregion to fit
	 * \param iterations number of iterations for solution to converge
	 * \param d_Parameters array of fitting parameters to return for each subregion
	 * \param d_CRLBs array of Cramer-Rao lower bound estimates to return for each subregion
	 * \param d_LogLikelihood array of loglikelihood estimates to return for each subregion
	 * \param Nfits number of subregions to fit
	 */
    
    //__shared__ float s_data[MEM];
    float M[NV_PS*NV_PS], Diag[NV_PS], Minv[NV_PS*NV_PS];
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int BlockSize = blockDim.x;
    int ii, jj, kk, ll;
    float model, cf, df, data;
    float Div;
    float PSFy, PSFx;
    int NV=NV_PS;
    float dudt[NV_PS];
    float d2udt2[NV_PS];
    float NR_Numerator[NV_PS], NR_Denominator[NV_PS];
    float theta[NV_PS];
    float maxjump[NV_PS]={1e0f, 1e0f, 1e2f, 2e0f, 5e-1f};
    float gamma[NV_PS]={1.0f, 1.0f, 0.5f, 1.0f, 1.0f};
    float Nmax;
    
    //Prevent read/write past end of array
    if ((bx*BlockSize+tx)>=Nfits) return;
    
	memset(M,0,NV_PS*NV_PS*sizeof(float));
	memset(Minv,0,NV_PS*NV_PS*sizeof(float));      
    //load data
    const float *s_data = d_data+(sz*sz*bx*BlockSize+sz*sz*tx);
    
    //initial values
    kernel_CenterofMass2D(sz, s_data, &theta[0], &theta[1]);
    kernel_GaussFMaxMin2D(sz, PSFSigma, s_data, &Nmax, &theta[3]);
    theta[2]=max(0.0f, (Nmax-theta[3])*2*pi*PSFSigma*PSFSigma);
    theta[4]=PSFSigma;
    
    for (kk=0;kk<iterations;kk++) {//main iterative loop
        
        //initialize
  		memset(NR_Numerator,0,NV_PS*sizeof(float));
		memset(NR_Denominator,0,NV_PS*sizeof(float));
      
        for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
            PSFx=kernel_IntGauss1D(ii, theta[0], theta[4]);
            PSFy=kernel_IntGauss1D(jj, theta[1], theta[4]);
            
            model=theta[3]+theta[2]*PSFx*PSFy;
            data=s_data[sz*jj+ii];
            
            //calculating derivatives
            kernel_DerivativeIntGauss1D(ii, theta[0], theta[4], theta[2], PSFy, &dudt[0], &d2udt2[0]);
            kernel_DerivativeIntGauss1D(jj, theta[1], theta[4], theta[2], PSFx, &dudt[1], &d2udt2[1]);
            kernel_DerivativeIntGauss2DSigma(ii, jj, theta[0], theta[1], theta[4], theta[2], PSFx, PSFy, &dudt[4], &d2udt2[4]);
            dudt[2] = PSFx*PSFy;
            d2udt2[2] = 0.0f;
            dudt[3] = 1.0f;
            d2udt2[3] = 0.0f;
            
            cf=0.0f;
            df=0.0f;
            if (model>10e-3f) cf=data/model-1;
            if (model>10e-3f) df=data/pow(model, 2);
            cf=min(cf, 10e4f);
            df=min(df, 10e4f);
            
            for (ll=0;ll<NV;ll++){
                NR_Numerator[ll]+=dudt[ll]*cf;
                NR_Denominator[ll]+=d2udt2[ll]*cf-pow(dudt[ll], 2)*df;
            }
        }
        
        // The update
        if (kk<5)
            for (ll=0;ll<NV;ll++)
                theta[ll]-=gamma[ll]*min(max(NR_Numerator[ll]/NR_Denominator[ll], -maxjump[ll]), maxjump[ll]);
        else
            for (ll=0;ll<NV;ll++)
                theta[ll]-=min(max(NR_Numerator[ll]/NR_Denominator[ll], -maxjump[ll]), maxjump[ll]);
        
        // Any other constraints
        theta[2]=max(theta[2], 1.0f);
        theta[3]=max(theta[3], 0.01f);
        theta[4]=max(theta[4], 0.5f);
        theta[4]=min(theta[4], sz/2.0f);
    }
    
    // Calculating the CRLB and LogLikelihood
    Div=0.0f;
    for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
        PSFx=kernel_IntGauss1D(ii, theta[0], PSFSigma);
        PSFy=kernel_IntGauss1D(jj, theta[1], PSFSigma);
        
        model=theta[3]+theta[2]*PSFx*PSFy;
        data=s_data[sz*jj+ii];
        
        //calculating derivatives
        kernel_DerivativeIntGauss1D(ii, theta[0], theta[4], theta[2], PSFy, &dudt[0], NULL);
        kernel_DerivativeIntGauss1D(jj, theta[1], theta[4], theta[2], PSFx, &dudt[1], NULL);
        kernel_DerivativeIntGauss2DSigma(ii, jj, theta[0], theta[1], theta[4], theta[2], PSFx, PSFy, &dudt[4], NULL);
        dudt[2] = PSFx*PSFy;
        dudt[3] = 1.0f;
        
        //Building the Fisher Information Matrix
        for (kk=0;kk<NV;kk++)for (ll=kk;ll<NV;ll++){
            M[kk*NV+ll]+= dudt[ll]*dudt[kk]/model;
            M[ll*NV+kk]=M[kk*NV+ll];
        }
        
        //LogLikelyhood
        if (model>0)
            if (data>0)Div+=data*log(model)-model-data*log(data)+data;
            else
                Div+=-model;
    }
    
    // Matrix inverse (CRLB=F^-1) and output assigments
    kernel_MatInvN(M, Minv, Diag, NV);
  
    
    //write to global arrays
    for (kk=0;kk<NV;kk++) d_Parameters[Nfits*kk+BlockSize*bx+tx]=theta[kk];
    for (kk=0;kk<NV;kk++) d_CRLBs[Nfits*kk+BlockSize*bx+tx]=Diag[kk];
    d_LogLikelihood[BlockSize*bx+tx] = Div;
    
    return;
}

//*******************************************************************************************
__global__ void kernel_MLEFit_z(const float *d_data, const float PSFSigma_x, const float Ax, const float Ay, const float Bx, 
	const float By, const float gamma, const float d, const float PSFSigma_y, const int sz, const int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,const int Nfits){
	/*! 
	 * \brief basic MLE fitting kernel.  No additional parameters are computed.
	 * \param d_data array of subregions to fit copied to GPU
	 * \param PSFSigma_x sigma of the point spread function on the x axis
	 * \param Ax ???
	 * \param Ay ???
	 * \param Bx ???
	 * \param By ???
	 * \param gamma ???
	 * \param d ???
	 * \param PSFSigma_y sigma of the point spread function on the y axis
	 * \param sz nxn size of the subregion to fit
	 * \param iterations number of iterations for solution to converge
	 * \param d_Parameters array of fitting parameters to return for each subregion
	 * \param d_CRLBs array of Cramer-Rao lower bound estimates to return for each subregion
	 * \param d_LogLikelihood array of loglikelihood estimates to return for each subregion
	 * \param Nfits number of subregions to fit
	 */
    //__shared__ float s_data[MEM];
    float M[5*5], Diag[5], Minv[5*5];
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int BlockSize = blockDim.x;
    int ii, jj, kk, ll;
    float model, cf, df, data;
    float Div;
    float PSFy, PSFx;
    int NV=5;
    float dudt[5];
    float d2udt2[5];
    float NR_Numerator[5], NR_Denominator[5];
    float theta[5];
    float maxjump[5]={1e0f, 1e0f, 1e2f, 2e0f, 1e-1f};
    float g[5]={1.0f, 1.0f, 0.5f, 1.0f, 1.0f};
    float Nmax;
    
    //Prevent read/write past end of array
    if ((bx*BlockSize+tx)>=Nfits) return;

	memset(M,0,NV*NV*sizeof(float));
	memset(Minv,0,NV*NV*sizeof(float));      
    
    //load data
    const float *s_data = d_data+(sz*sz*bx*BlockSize+sz*sz*tx);

    //initial values
    kernel_CenterofMass2D(sz, s_data, &theta[0], &theta[1]);
    kernel_GaussFMaxMin2D(sz, PSFSigma_x, s_data, &Nmax, &theta[3]);
    theta[2]=max(0.0f, (Nmax-theta[3])*2*pi*PSFSigma_x*PSFSigma_y*sqrt(2.0f));
    theta[4]=0;
   
    for (kk=0;kk<iterations;kk++) {//main iterative loop
        
        //initialize
  		memset(NR_Numerator,0,NV*sizeof(float));
		memset(NR_Denominator,0,NV*sizeof(float));
        
        for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
            kernel_DerivativeIntGauss2Dz(ii, jj, theta, PSFSigma_x,PSFSigma_y, Ax,Ay,Bx,By, gamma, d, &PSFx, &PSFy, dudt, d2udt2);
            
            model=theta[3]+theta[2]*PSFx*PSFy;
            data=s_data[sz*jj+ii];
            
            //calculating remaining derivatives
            dudt[2] = PSFx*PSFy;
            d2udt2[2] = 0.0f;
            dudt[3] = 1.0f;
            d2udt2[3] = 0.0f;
            
            cf=0.0f;
            df=0.0f;
            if (model>10e-3f) cf=data/model-1;
            if (model>10e-3f) df=data/pow(model, 2);
            cf=min(cf, 10e4f);
            df=min(df, 10e4f);
            
            for (ll=0;ll<NV;ll++){
                NR_Numerator[ll]+=dudt[ll]*cf;
                NR_Denominator[ll]+=d2udt2[ll]*cf-pow(dudt[ll], 2)*df;
            }
        }
        
        // The update
        if (kk<2)
            for (ll=0;ll<NV;ll++)
                theta[ll]-=g[ll]*min(max(NR_Numerator[ll]/NR_Denominator[ll], -maxjump[ll]), maxjump[ll]);
        else
            for (ll=0;ll<NV;ll++)
                theta[ll]-=min(max(NR_Numerator[ll]/NR_Denominator[ll], -maxjump[ll]), maxjump[ll]);
        
        // Any other constraints
        theta[2]=max(theta[2], 1.0f);
        theta[3]=max(theta[3], 0.01f);
        
    }
    
    // Calculating the CRLB and LogLikelihood
    Div=0.0f;
    for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
        
        kernel_DerivativeIntGauss2Dz(ii, jj, theta, PSFSigma_x,PSFSigma_y, Ax,Ay, Bx,By, gamma, d, &PSFx, &PSFy, dudt, NULL);
        
        model=theta[3]+theta[2]*PSFx*PSFy;
        data=s_data[sz*jj+ii];
        
        //calculating remaining derivatives
        dudt[2] = PSFx*PSFy;
        dudt[3] = 1.0f;
       
        //Building the Fisher Information Matrix
        for (kk=0;kk<NV;kk++)for (ll=kk;ll<NV;ll++){
            M[kk*NV+ll]+= dudt[ll]*dudt[kk]/model;
            M[ll*NV+kk]=M[kk*NV+ll];
        }
        
        //LogLikelyhood
        if (model>0)
            if (data>0)Div+=data*log(model)-model-data*log(data)+data;
            else
                Div+=-model;
    }
    
    // Matrix inverse (CRLB=F^-1) 
    kernel_MatInvN(M, Minv, Diag, NV);
  
   //write to global arrays
    for (kk=0;kk<NV;kk++) d_Parameters[Nfits*kk+BlockSize*bx+tx]=theta[kk];
    for (kk=0;kk<NV;kk++) d_CRLBs[Nfits*kk+BlockSize*bx+tx]=Diag[kk];
    d_LogLikelihood[BlockSize*bx+tx] = Div;
    return;
}

//*******************************************************************************************
__global__ void kernel_MLEFit_sigmaxy(const float *d_data, const float PSFSigma, const int sz, const int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,const int Nfits){
	/*! 
	 * \brief basic MLE fitting kernel.  No additional parameters are computed.
	 * \param d_data array of subregions to fit copied to GPU
	 * \param PSFSigma sigma of the point spread function
	 * \param sz nxn size of the subregion to fit
	 * \param iterations number of iterations for solution to converge
	 * \param d_Parameters array of fitting parameters to return for each subregion
	 * \param d_CRLBs array of Cramer-Rao lower bound estimates to return for each subregion
	 * \param d_LogLikelihood array of loglikelihood estimates to return for each subregion
	 * \param Nfits number of subregions to fit
	 */
 
    //__shared__ float s_data[MEM];
    float M[6*6], Diag[6], Minv[6*6];
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int BlockSize = blockDim.x;
    int ii, jj, kk, ll;
    float model, cf, df, data;
    float Div;
    float PSFy, PSFx;
    int NV=6;
    float dudt[6];
    float d2udt2[6];
    float NR_Numerator[6], NR_Denominator[6];
    float theta[6];
    float maxjump[6]={1e0f, 1e0f, 1e2f, 2e0f, 1e-1f,1e-1f};
    float g[6]={1.0f, 1.0f, 0.5f, 1.0f, 1.0f,1.0f};
    float Nmax;
    
    //Prevent read/write past end of array
    if ((bx*BlockSize+tx)>=Nfits) return;
    
	memset(M,0,NV*NV*sizeof(float));
	memset(Minv,0,NV*NV*sizeof(float));      
    
    //load data
    const float *s_data = d_data+(sz*sz*bx*BlockSize+sz*sz*tx);
    
	//initial values
    kernel_CenterofMass2D(sz, s_data, &theta[0], &theta[1]);
    kernel_GaussFMaxMin2D(sz, PSFSigma, s_data, &Nmax, &theta[3]);
    theta[2]=max(0.0f, (Nmax-theta[3])*2*pi*PSFSigma*PSFSigma);
    theta[4]=PSFSigma;
    theta[5]=PSFSigma;
    for (kk=0;kk<iterations;kk++) {//main iterative loop
        
        //initialize
  		memset(NR_Numerator,0,NV*sizeof(float));
		memset(NR_Denominator,0,NV*sizeof(float));
        
        for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
            PSFx=kernel_IntGauss1D(ii, theta[0], theta[4]);
            PSFy=kernel_IntGauss1D(jj, theta[1], theta[5]);
            
            model=theta[3]+theta[2]*PSFx*PSFy;
            data=s_data[sz*jj+ii];
            
            //calculating derivatives
            kernel_DerivativeIntGauss1D(ii, theta[0], theta[4], theta[2], PSFy, &dudt[0], &d2udt2[0]);
            kernel_DerivativeIntGauss1D(jj, theta[1], theta[5], theta[2], PSFx, &dudt[1], &d2udt2[1]);
            kernel_DerivativeIntGauss1DSigma(ii, theta[0], theta[4], theta[2], PSFy, &dudt[4], &d2udt2[4]);
            kernel_DerivativeIntGauss1DSigma(jj, theta[1], theta[5], theta[2], PSFx, &dudt[5], &d2udt2[5]);
            dudt[2] = PSFx*PSFy;
            d2udt2[2] = 0.0f;
            dudt[3] = 1.0f;
            d2udt2[3] = 0.0f;
            
            cf=0.0f;
            df=0.0f;
            if (model>10e-3f) cf=data/model-1;
            if (model>10e-3f) df=data/pow(model, 2);
            cf=min(cf, 10e4f);
            df=min(df, 10e4f);
            
            for (ll=0;ll<NV;ll++){
                NR_Numerator[ll]+=dudt[ll]*cf;
                NR_Denominator[ll]+=d2udt2[ll]*cf-pow(dudt[ll], 2)*df;
            }
        }
        
        // The update
            for (ll=0;ll<NV;ll++)
                theta[ll]-=g[ll]*min(max(NR_Numerator[ll]/NR_Denominator[ll], -maxjump[ll]), maxjump[ll]);

        // Any other constraints
        theta[2]=max(theta[2], 1.0f);
        theta[3]=max(theta[3], 0.01f);
        theta[4]=max(theta[4], PSFSigma/10.0f);
        theta[5]=max(theta[5], PSFSigma/10.0f);  
    }
    
    // Calculating the CRLB and LogLikelihood
    Div=0.0f;
    for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
        
        PSFx=kernel_IntGauss1D(ii, theta[0], theta[4]);
        PSFy=kernel_IntGauss1D(jj, theta[1], theta[5]);
        
        model=theta[3]+theta[2]*PSFx*PSFy;
        data=s_data[sz*jj+ii];
        
        //calculating derivatives
        kernel_DerivativeIntGauss1D(ii, theta[0], theta[4], theta[2], PSFy, &dudt[0], NULL);
        kernel_DerivativeIntGauss1D(jj, theta[1], theta[5], theta[2], PSFx, &dudt[1], NULL);
        kernel_DerivativeIntGauss1DSigma(ii, theta[0], theta[4], theta[2], PSFy, &dudt[4], NULL);
        kernel_DerivativeIntGauss1DSigma(jj, theta[1], theta[5], theta[2], PSFx, &dudt[5], NULL);
        dudt[2] = PSFx*PSFy;
        dudt[3] = 1.0f;
        
        //Building the Fisher Information Matrix
        for (kk=0;kk<NV;kk++)for (ll=kk;ll<NV;ll++){
            M[kk*NV+ll]+= dudt[ll]*dudt[kk]/model;
            M[ll*NV+kk]=M[kk*NV+ll];
        }
        
        //LogLikelyhood
        if (model>0)
            if (data>0)Div+=data*log(model)-model-data*log(data)+data;
            else
                Div+=-model;
    }
    
    // Matrix inverse (CRLB=F^-1) and output assigments
    kernel_MatInvN(M, Minv, Diag, NV);
   
    //write to global arrays
    for (kk=0;kk<NV;kk++) d_Parameters[Nfits*kk+BlockSize*bx+tx]=theta[kk];
    for (kk=0;kk<NV;kk++) d_CRLBs[Nfits*kk+BlockSize*bx+tx]=Diag[kk];
    d_LogLikelihood[BlockSize*bx+tx] = Div;
    return;
}
