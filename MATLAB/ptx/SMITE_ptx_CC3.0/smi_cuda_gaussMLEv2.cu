#include "hip/hip_runtime.h"
/*!
 * \file GPUgaussMLEv2.cu
 * \author Keith Lidke
 * \date January 10, 2010
 * \brief This file contains all of the Cuda kernels.  The helper functions
 * are defined in GPUgaussLib.cuh
 */

#include "definitions.h"
#include "MatInvLib.h"
#include "GPUgaussLib.cuh"
#include "GPUgaussMLEv2.h"

//*******************************************************************************************
//theta is: {x,y,N,bg}
__global__ void kernel_MLEFit_XYNB_(const float *d_data, const float PSFSigma, const int sz, const int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,const int Nfits){
	/*! 
	 * \brief basic MLE fitting kernel.  No additional parameters are computed.
	 * \param d_data array of subregions to fit copied to GPU
	 * \param PSFSigma sigma of the point spread function
	 * \param sz nxn size of the subregion to fit
	 * \param iterations number of iterations for solution to converge
	 * \param d_Parameters array of fitting parameters to return for each subregion
	 * \param d_CRLBs array of Cramer-Rao lower bound estimates to return for each subregion
	 * \param d_LogLikelihood array of loglikelihood estimates to return for each subregion
	 * \param Nfits number of subregions to fit
	 */
    //__shared__ float s_data[MEM];
    float M[NV_P*NV_P], Diag[NV_P], Minv[NV_P*NV_P];
    const int tx = threadIdx.x;
    const int bx = blockIdx.x;
    const int BlockSize = blockDim.x;
    int ii, jj, kk, ll;
    float model, cf, df, data;
    float Div;
    float PSFy, PSFx;
    int NV=NV_P;
    float dudt[NV_P];
    float d2udt2[NV_P];
    float NR_Numerator[NV_P], NR_Denominator[NV_P];
    float theta[NV_P];
    float maxjump[NV_P]={1e0f, 1e0f, 1e2f, 2e0f};
    float Nmax;

    //Prevent read/write past end of array
    if ((bx*BlockSize+tx)>=Nfits) return;
    
	memset(M,0,NV_P*NV_P*sizeof(float));
	memset(Minv,0,NV_P*NV_P*sizeof(float));
    //load data
    const float *s_data = d_data+(sz*sz*bx*BlockSize+sz*sz*tx);
    //initial values
    kernel_CenterofMass2D(sz, s_data, &theta[0], &theta[1]);
    kernel_GaussFMaxMin2D(sz, PSFSigma, s_data, &Nmax, &theta[3]);
    theta[2]=max(0.0f, (Nmax-theta[3])*2*2*pi*PSFSigma*PSFSigma); //Added 2* on 8.9.16 to account for smoothing filter.
    
    for (kk=0;kk<iterations;kk++) {//main iterative loop
        
        //initialize
		memset(NR_Numerator,0,NV_P*sizeof(float));
		memset(NR_Denominator,0,NV_P*sizeof(float));

        for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
            PSFx=kernel_IntGauss1D(ii, theta[0], PSFSigma);
            PSFy=kernel_IntGauss1D(jj, theta[1], PSFSigma);
            
            model=theta[3]+theta[2]*PSFx*PSFy;
            data=s_data[sz*jj+ii];
            
            //calculating derivatives
            kernel_DerivativeIntGauss1D(ii, theta[0], PSFSigma, theta[2], PSFy, &dudt[0], &d2udt2[0]);
            kernel_DerivativeIntGauss1D(jj, theta[1], PSFSigma, theta[2], PSFx, &dudt[1], &d2udt2[1]);
            dudt[2] = PSFx*PSFy;
            d2udt2[2] = 0.0f;
            dudt[3] = 1.0f;
            d2udt2[3] = 0.0f;
            
            cf=0.0f;
            df=0.0f;
            if (model>10e-3f) cf=data/model-1;
            if (model>10e-3f) df=data/pow(model, 2);
            cf=min(cf, 10e4f);
            df=min(df, 10e4f);
            
            for (ll=0;ll<NV;ll++){
                NR_Numerator[ll]+=dudt[ll]*cf;
                NR_Denominator[ll]+=d2udt2[ll]*cf-pow(dudt[ll], 2)*df;
            }
        }
         
        // The update: Editted 8.9.16 to allow for larger jumps in Photons/BG
        theta[0]-=min(max(NR_Numerator[0]/NR_Denominator[0], -maxjump[0]), maxjump[0]);
        theta[1]-=min(max(NR_Numerator[1]/NR_Denominator[1], -maxjump[1]), maxjump[1]);
        theta[2]-=min(max(NR_Numerator[2]/NR_Denominator[2], -theta[2]), theta[2]);
        theta[3]-=min(max(NR_Numerator[3]/NR_Denominator[3], -theta[3]), theta[3]);

        // Any other constraints
        theta[2]=max(theta[2], 1.0f); //Make sure Photons is postitve
        theta[3]=max(theta[3], 0.01f); //Make sure Background is postitve
        
    }
    
    // Calculating the CRLB and LogLikelihood
    Div=0.0;
    for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
        PSFx=kernel_IntGauss1D(ii, theta[0], PSFSigma);
        PSFy=kernel_IntGauss1D(jj, theta[1], PSFSigma);
        
        model=theta[3]+theta[2]*PSFx*PSFy;
        data=s_data[sz*jj+ii];
        
        //calculating derivatives
        kernel_DerivativeIntGauss1D(ii, theta[0], PSFSigma, theta[2], PSFy, &dudt[0], NULL);
        kernel_DerivativeIntGauss1D(jj, theta[1], PSFSigma, theta[2], PSFx, &dudt[1], NULL);
        dudt[2] = PSFx*PSFy;
        dudt[3] = 1.0f;
        
        //Building the Fisher Information Matrix
        for (kk=0;kk<NV;kk++)for (ll=kk;ll<NV;ll++){
            M[kk*NV+ll]+= dudt[ll]*dudt[kk]/model;
            M[ll*NV+kk]=M[kk*NV+ll];
        }
        
        //LogLikelyhood
        if (model>0)
            if (data>0)Div+=data*log(model)-model-data*log(data)+data;
            else
                Div+=-model;
    }
    
    // Matrix inverse (CRLB=F^-1) and output assigments
    kernel_MatInvN(M, Minv, Diag, NV);
    
    //write to global arrays
    for (kk=0;kk<NV;kk++) d_Parameters[Nfits*kk+BlockSize*bx+tx]=theta[kk];
    for (kk=0;kk<NV;kk++) d_CRLBs[Nfits*kk+BlockSize*bx+tx]=Diag[kk];
    d_LogLikelihood[BlockSize*bx+tx] = Div;
    
    return;
}

//*******************************************************************************************
__global__ void kernel_MLEFit_XYNBS_(const float *d_data, const float PSFSigma, const int sz, const int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,const int Nfits){
	/*! 
	 * \brief basic MLE fitting kernel.  No additional parameters are computed.
	 * \param d_data array of subregions to fit copied to GPU
	 * \param PSFSigma sigma of the point spread function
	 * \param sz nxn size of the subregion to fit
	 * \param iterations number of iterations for solution to converge
	 * \param d_Parameters array of fitting parameters to return for each subregion
	 * \param d_CRLBs array of Cramer-Rao lower bound estimates to return for each subregion
	 * \param d_LogLikelihood array of loglikelihood estimates to return for each subregion
	 * \param Nfits number of subregions to fit
	 */
    
    //__shared__ float s_data[MEM];
    float M[NV_PS*NV_PS], Diag[NV_PS], Minv[NV_PS*NV_PS];
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int BlockSize = blockDim.x;
    int ii, jj, kk, ll;
    float model, cf, df, data;
    float Div;
    float PSFy, PSFx;
    int NV=NV_PS;
    float dudt[NV_PS];
    float d2udt2[NV_PS];
    float NR_Numerator[NV_PS], NR_Denominator[NV_PS];
    float theta[NV_PS];
    float maxjump[NV_PS]={1e0f, 1e0f, 1e2f, 2e0f, 5e-1f};
    float Nmax;
    
    //Prevent read/write past end of array
    if ((bx*BlockSize+tx)>=Nfits) return;
    
	memset(M,0,NV_PS*NV_PS*sizeof(float));
	memset(Minv,0,NV_PS*NV_PS*sizeof(float));      
    //load data
    const float *s_data = d_data+(sz*sz*bx*BlockSize+sz*sz*tx);
    
    //initial values
    kernel_CenterofMass2D(sz, s_data, &theta[0], &theta[1]);
    kernel_GaussFMaxMin2D(sz, PSFSigma, s_data, &Nmax, &theta[3]);
    theta[2]=max(0.0f, (Nmax-theta[3])*2*2*pi*PSFSigma*PSFSigma);
    theta[4]=PSFSigma;
    
    for (kk=0;kk<iterations;kk++) {//main iterative loop
        
        //initialize
  		memset(NR_Numerator,0,NV_PS*sizeof(float));
		memset(NR_Denominator,0,NV_PS*sizeof(float));
      
        for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
            PSFx=kernel_IntGauss1D(ii, theta[0], theta[4]);
            PSFy=kernel_IntGauss1D(jj, theta[1], theta[4]);
            
            model=theta[3]+theta[2]*PSFx*PSFy;
            data=s_data[sz*jj+ii];
            
            //calculating derivatives
            kernel_DerivativeIntGauss1D(ii, theta[0], theta[4], theta[2], PSFy, &dudt[0], &d2udt2[0]);
            kernel_DerivativeIntGauss1D(jj, theta[1], theta[4], theta[2], PSFx, &dudt[1], &d2udt2[1]);
            kernel_DerivativeIntGauss2DSigma(ii, jj, theta[0], theta[1], theta[4], theta[2], PSFx, PSFy, &dudt[4], &d2udt2[4]);
            dudt[2] = PSFx*PSFy;
            d2udt2[2] = 0.0f;
            dudt[3] = 1.0f;
            d2udt2[3] = 0.0f;
            
            cf=0.0f;
            df=0.0f;
            if (model>10e-3f) cf=data/model-1;
            if (model>10e-3f) df=data/pow(model, 2);
            cf=min(cf, 10e4f);
            df=min(df, 10e4f);
            
            for (ll=0;ll<NV;ll++){
                NR_Numerator[ll]+=dudt[ll]*cf;
                NR_Denominator[ll]+=d2udt2[ll]*cf-pow(dudt[ll], 2)*df;
            }
        }
        
        // The update: Editted 8.9.16 to allow for larger jumps in Photons/BG
        theta[0]-=min(max(NR_Numerator[0]/NR_Denominator[0], -maxjump[0]), maxjump[0]);
        theta[1]-=min(max(NR_Numerator[1]/NR_Denominator[1], -maxjump[1]), maxjump[1]);
        theta[2]-=min(max(NR_Numerator[2]/NR_Denominator[2], -theta[2]), theta[2]);
        theta[3]-=min(max(NR_Numerator[3]/NR_Denominator[3], -theta[3]), theta[3]);
        theta[4]-=min(max(NR_Numerator[4]/NR_Denominator[4], -theta[4]), theta[4]);

        // Any other constraints
        theta[2]=max(theta[2], 1.0f); //Make sure Photons is postitve
        theta[3]=max(theta[3], 0.01f); //Make sure Background is postitve
        theta[4]=max(theta[4], 0.5f); //Constrain Sigma
        theta[4]=min(theta[4], sz/2.0f); //Constrain Sigma
    }
    
    // Calculating the CRLB and LogLikelihood
    Div=0.0f;
    for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
        PSFx=kernel_IntGauss1D(ii, theta[0], theta[4]); //bug fix 8.9.16 
        PSFy=kernel_IntGauss1D(jj, theta[1], theta[4]); //bug fix 8.9.16 
        
        model=theta[3]+theta[2]*PSFx*PSFy;
        data=s_data[sz*jj+ii];
        
        //calculating derivatives
        kernel_DerivativeIntGauss1D(ii, theta[0], theta[4], theta[2], PSFy, &dudt[0], NULL);
        kernel_DerivativeIntGauss1D(jj, theta[1], theta[4], theta[2], PSFx, &dudt[1], NULL);
        kernel_DerivativeIntGauss2DSigma(ii, jj, theta[0], theta[1], theta[4], theta[2], PSFx, PSFy, &dudt[4], NULL);
        dudt[2] = PSFx*PSFy;
        dudt[3] = 1.0f;
        
        //Building the Fisher Information Matrix
        for (kk=0;kk<NV;kk++)for (ll=kk;ll<NV;ll++){
            M[kk*NV+ll]+= dudt[ll]*dudt[kk]/model;
            M[ll*NV+kk]=M[kk*NV+ll];
        }
        
        //LogLikelyhood
        if (model>0)
            if (data>0)Div+=data*log(model)-model-data*log(data)+data;
            else
                Div+=-model;
    }
    
    // Matrix inverse (CRLB=F^-1) and output assigments
    kernel_MatInvN(M, Minv, Diag, NV);
  
    
    //write to global arrays
    for (kk=0;kk<NV;kk++) d_Parameters[Nfits*kk+BlockSize*bx+tx]=theta[kk];
    for (kk=0;kk<NV;kk++) d_CRLBs[Nfits*kk+BlockSize*bx+tx]=Diag[kk];
    d_LogLikelihood[BlockSize*bx+tx] = Div;
    
    return;
}

//*******************************************************************************************
__global__ void kernel_MLEFit_XYNBZ_(const float *d_data, const float PSFSigma_x, const float Ax, const float Ay, const float Bx, 
	const float By, const float gamma, const float d, const float PSFSigma_y, const int sz, const int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,const int Nfits){
	/*! 
	 * \brief basic MLE fitting kernel.  No additional parameters are computed.
	 * \param d_data array of subregions to fit copied to GPU
	 * \param PSFSigma_x sigma of the point spread function on the x axis
	 * \param Ax ???
	 * \param Ay ???
	 * \param Bx ???
	 * \param By ???
	 * \param gamma ???
	 * \param d ???
	 * \param PSFSigma_y sigma of the point spread function on the y axis
	 * \param sz nxn size of the subregion to fit
	 * \param iterations number of iterations for solution to converge
	 * \param d_Parameters array of fitting parameters to return for each subregion
	 * \param d_CRLBs array of Cramer-Rao lower bound estimates to return for each subregion
	 * \param d_LogLikelihood array of loglikelihood estimates to return for each subregion
	 * \param Nfits number of subregions to fit
	 */
    //__shared__ float s_data[MEM];
    float M[5*5], Diag[5], Minv[5*5];
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int BlockSize = blockDim.x;
    int ii, jj, kk, ll;
    float model, cf, df, data;
    float Div;
    float PSFy, PSFx;
    int NV=5;
    float dudt[5];
    float d2udt2[5];
    float NR_Numerator[5], NR_Denominator[5];
    float theta[5];
    float maxjump[5]={1e0f, 1e0f, 1e2f, 2e0f, 1e-1f};
    float Nmax;
    
    //Prevent read/write past end of array
    if ((bx*BlockSize+tx)>=Nfits) return;

	memset(M,0,NV*NV*sizeof(float));
	memset(Minv,0,NV*NV*sizeof(float));      
    
    //load data
    const float *s_data = d_data+(sz*sz*bx*BlockSize+sz*sz*tx);

    //initial values
    kernel_CenterofMass2D(sz, s_data, &theta[0], &theta[1]);
    kernel_GaussFMaxMin2D(sz, PSFSigma_x, s_data, &Nmax, &theta[3]);
    theta[2]=max(0.0f, (Nmax-theta[3])*2*pi*PSFSigma_x*PSFSigma_y*sqrt(2.0f));
    theta[4]=0;
   
    for (kk=0;kk<iterations;kk++) {//main iterative loop
        
        //initialize
  		memset(NR_Numerator,0,NV*sizeof(float));
		memset(NR_Denominator,0,NV*sizeof(float));
        
        for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
            kernel_DerivativeIntGauss2Dz(ii, jj, theta, PSFSigma_x,PSFSigma_y, Ax,Ay,Bx,By, gamma, d, &PSFx, &PSFy, dudt, d2udt2);
            
            model=theta[3]+theta[2]*PSFx*PSFy;
            data=s_data[sz*jj+ii];
            
            //calculating remaining derivatives
            dudt[2] = PSFx*PSFy;
            d2udt2[2] = 0.0f;
            dudt[3] = 1.0f;
            d2udt2[3] = 0.0f;
            
            cf=0.0f;
            df=0.0f;
            if (model>10e-3f) cf=data/model-1;
            if (model>10e-3f) df=data/pow(model, 2);
            cf=min(cf, 10e4f);
            df=min(df, 10e4f);
            
            for (ll=0;ll<NV;ll++){
                NR_Numerator[ll]+=dudt[ll]*cf;
                NR_Denominator[ll]+=d2udt2[ll]*cf-pow(dudt[ll], 2)*df;
            }
        }
        
        // The update: Editted 8.9.16 to allow for larger jumps in Photons/BG
        theta[0]-=min(max(NR_Numerator[0]/NR_Denominator[0], -maxjump[0]), maxjump[0]);
        theta[1]-=min(max(NR_Numerator[1]/NR_Denominator[1], -maxjump[1]), maxjump[1]);
        theta[2]-=min(max(NR_Numerator[2]/NR_Denominator[2], -theta[2]), theta[2]);
        theta[3]-=min(max(NR_Numerator[3]/NR_Denominator[3], -theta[3]), theta[3]);
        theta[4]-=min(max(NR_Numerator[4]/NR_Denominator[4], -maxjump[4]), maxjump[4]);
        
        // Any other constraints
        theta[2]=max(theta[2], 1.0f);
        theta[3]=max(theta[3], 0.01f);
        
    }
    
    // Calculating the CRLB and LogLikelihood
    Div=0.0f;
    for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
        
        kernel_DerivativeIntGauss2Dz(ii, jj, theta, PSFSigma_x,PSFSigma_y, Ax,Ay, Bx,By, gamma, d, &PSFx, &PSFy, dudt, NULL);
        
        model=theta[3]+theta[2]*PSFx*PSFy;
        data=s_data[sz*jj+ii];
        
        //calculating remaining derivatives
        dudt[2] = PSFx*PSFy;
        dudt[3] = 1.0f;
       
        //Building the Fisher Information Matrix
        for (kk=0;kk<NV;kk++)for (ll=kk;ll<NV;ll++){
            M[kk*NV+ll]+= dudt[ll]*dudt[kk]/model;
            M[ll*NV+kk]=M[kk*NV+ll];
        }
        
        //LogLikelyhood
        if (model>0)
            if (data>0)Div+=data*log(model)-model-data*log(data)+data;
            else
                Div+=-model;
    }
    
    // Matrix inverse (CRLB=F^-1) 
    kernel_MatInvN(M, Minv, Diag, NV);
  
   //write to global arrays
    for (kk=0;kk<NV;kk++) d_Parameters[Nfits*kk+BlockSize*bx+tx]=theta[kk];
    for (kk=0;kk<NV;kk++) d_CRLBs[Nfits*kk+BlockSize*bx+tx]=Diag[kk];
    d_LogLikelihood[BlockSize*bx+tx] = Div;
    return;
}

//*******************************************************************************************
__global__ void kernel_MLEFit_XYNBSXSY_(const float *d_data, const float PSFSigma, const int sz, const int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,const int Nfits){
	/*! 
	 * \brief basic MLE fitting kernel.  No additional parameters are computed.
	 * \param d_data array of subregions to fit copied to GPU
	 * \param PSFSigma sigma of the point spread function
	 * \param sz nxn size of the subregion to fit
	 * \param iterations number of iterations for solution to converge
	 * \param d_Parameters array of fitting parameters to return for each subregion
	 * \param d_CRLBs array of Cramer-Rao lower bound estimates to return for each subregion
	 * \param d_LogLikelihood array of loglikelihood estimates to return for each subregion
	 * \param Nfits number of subregions to fit
	 */
 
    //__shared__ float s_data[MEM];
    float M[6*6], Diag[6], Minv[6*6];
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int BlockSize = blockDim.x;
    int ii, jj, kk, ll;
    float model, cf, df, data;
    float Div;
    float PSFy, PSFx;
    int NV=6;
    float dudt[6];
    float d2udt2[6];
    float NR_Numerator[6], NR_Denominator[6];
    float theta[6];
    float maxjump[6]={1e0f, 1e0f, 1e2f, 2e0f, 1e-1f,1e-1f};
    float Nmax;
    
    //Prevent read/write past end of array
    if ((bx*BlockSize+tx)>=Nfits) return;
    
	memset(M,0,NV*NV*sizeof(float));
	memset(Minv,0,NV*NV*sizeof(float));      
    
    //load data
    const float *s_data = d_data+(sz*sz*bx*BlockSize+sz*sz*tx);
    
	//initial values
    kernel_CenterofMass2D(sz, s_data, &theta[0], &theta[1]);
    kernel_GaussFMaxMin2D(sz, PSFSigma, s_data, &Nmax, &theta[3]);
    theta[2]=max(0.0f, (Nmax-theta[3])*2*2*pi*PSFSigma*PSFSigma);
    theta[4]=PSFSigma;
    theta[5]=PSFSigma;
    for (kk=0;kk<iterations;kk++) {//main iterative loop
        
        //initialize
  		memset(NR_Numerator,0,NV*sizeof(float));
		memset(NR_Denominator,0,NV*sizeof(float));
        
        for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
            PSFx=kernel_IntGauss1D(ii, theta[0], theta[4]);
            PSFy=kernel_IntGauss1D(jj, theta[1], theta[5]);
            
            model=theta[3]+theta[2]*PSFx*PSFy;
            data=s_data[sz*jj+ii];
            
            //calculating derivatives
   
            kernel_DerivativeIntGauss1D(ii, theta[0], theta[4], theta[2], PSFy, &dudt[0], &d2udt2[0]);
            kernel_DerivativeIntGauss1D(jj, theta[1], theta[5], theta[2], PSFx, &dudt[1], &d2udt2[1]);
            kernel_DerivativeIntGauss1DSigma(ii, theta[0], theta[4], theta[2], PSFy, &dudt[4], &d2udt2[4]);
            kernel_DerivativeIntGauss1DSigma(jj, theta[1], theta[5], theta[2], PSFx, &dudt[5], &d2udt2[5]);
            
            
            dudt[2] = PSFx*PSFy;
            d2udt2[2] = 0.0f;
            dudt[3] = 1.0f;
            d2udt2[3] = 0.0f;
            
            cf=0.0f;
            df=0.0f;
            if (model>10e-3f) cf=data/model-1;
            if (model>10e-3f) df=data/pow(model, 2);
            cf=min(cf, 10e4f);
            df=min(df, 10e4f);
            
            for (ll=0;ll<NV;ll++){
                NR_Numerator[ll]+=dudt[ll]*cf;
                NR_Denominator[ll]+=d2udt2[ll]*cf-pow(dudt[ll], 2)*df;
            }
        }
        
         // The update: Editted 8.9.16 to allow for larger jumps in Photons/BG
        theta[0]-=min(max(NR_Numerator[0]/NR_Denominator[0], -maxjump[0]), maxjump[0]);
        theta[1]-=min(max(NR_Numerator[1]/NR_Denominator[1], -maxjump[1]), maxjump[1]);
        theta[2]-=min(max(NR_Numerator[2]/NR_Denominator[2], -theta[2]), theta[2]);
        theta[3]-=min(max(NR_Numerator[3]/NR_Denominator[3], -theta[3]), theta[3]);
        theta[4]-=min(max(NR_Numerator[4]/NR_Denominator[4], -theta[4]), theta[4]);
        theta[5]-=min(max(NR_Numerator[5]/NR_Denominator[5], -theta[5]), theta[5]);

        // Any other constraints
        theta[2]=max(theta[2], 1.0f); //Make sure Photons is postitve
        theta[3]=max(theta[3], 0.01f); //Make sure Background is postitve
        theta[4]=max(theta[4], 0.5f); //Constrain Sigma
        theta[4]=min(theta[4], sz/2.0f); //Constrain SigmaX
        theta[5]=max(theta[5], 0.5f); //Constrain Sigma
        theta[5]=min(theta[5], sz/2.0f); //Constrain SigmaX
    }
    
    // Calculating the CRLB and LogLikelihood
    Div=0.0f;
    for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
        
        PSFx=kernel_IntGauss1D(ii, theta[0], theta[4]);
        PSFy=kernel_IntGauss1D(jj, theta[1], theta[5]);
        
        model=theta[3]+theta[2]*PSFx*PSFy;
        data=s_data[sz*jj+ii];
        
        //calculating derivatives
        kernel_DerivativeIntGauss1D(ii, theta[0], theta[4], theta[2], PSFy, &dudt[0], NULL);
        kernel_DerivativeIntGauss1D(jj, theta[1], theta[5], theta[2], PSFx, &dudt[1], NULL);
        kernel_DerivativeIntGauss1DSigma(ii, theta[0], theta[4], theta[2], PSFy, &dudt[4], NULL);
        kernel_DerivativeIntGauss1DSigma(jj, theta[1], theta[5], theta[2], PSFx, &dudt[5], NULL);
        dudt[2] = PSFx*PSFy;
        dudt[3] = 1.0f;
        
        //Building the Fisher Information Matrix
        for (kk=0;kk<NV;kk++)for (ll=kk;ll<NV;ll++){
            M[kk*NV+ll]+= dudt[ll]*dudt[kk]/model;
            M[ll*NV+kk]=M[kk*NV+ll];
        }
        
        //LogLikelyhood
        if (model>0)
            if (data>0)Div+=data*log(model)-model-data*log(data)+data;
            else
                Div+=-model;
    }
    
    // Matrix inverse (CRLB=F^-1) and output assigments
    kernel_MatInvN(M, Minv, Diag, NV);
   
    //write to global arrays
    for (kk=0;kk<NV;kk++) d_Parameters[Nfits*kk+BlockSize*bx+tx]=theta[kk];
    for (kk=0;kk<NV;kk++) d_CRLBs[Nfits*kk+BlockSize*bx+tx]=Diag[kk];
    d_LogLikelihood[BlockSize*bx+tx] = Div;
    return;
}

// SCMOS Versions---------------------------------------

__global__ void kernel_MLEFit_SCMOSXYNB_(const float *d_data, const float *d_Coords, const float *d_GainRatio, 
	    const float PSFSigma, const int sz, const int Mapsz, const int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,const int Nfits){
	/*! 
	 * \brief basic MLE fitting kernel.  No additional parameters are computed.
	 * \param d_data array of subregions to fit copied to GPU
	 * \param d_Coords array of subregions's pixel coordinates in original field of view.
	 * \param d_GainRatio calibration result of variance/gain^2 in each pixel of original field of view. 
	 * \param PSFSigma sigma of the point spread function
	 * \param sz nxn size of the subregion to fit
	 * \param Mapsz size of original field of view.
	 * \param iterations number of iterations for solution to converge
	 * \param d_Parameters array of fitting parameters to return for each subregion
	 * \param d_CRLBs array of Cramer-Rao lower bound estimates to return for each subregion
	 * \param d_LogLikelihood array of loglikelihood estimates to return for each subregion
	 * \param Nfits number of subregions to fit
	 */
    //__shared__ float s_data[MEM];
    float M[NV_P*NV_P], Diag[NV_P], Minv[NV_P*NV_P];
    const int tx = threadIdx.x;
    const int bx = blockIdx.x;
    const int BlockSize = blockDim.x;
    int ii, jj, kk, ll;
    float model, cf, df, data;
    float Div;
    float PSFy, PSFx;
    int NV=NV_P;
    float dudt[NV_P];
    float d2udt2[NV_P];
    float NR_Numerator[NV_P], NR_Denominator[NV_P];
    float theta[NV_P];
    float maxjump[NV_P]={1e0f, 1e0f, 1e2f, 2e0f};
    float Nmax;
	float gainR;
	int GRind;
    //Prevent read/write past end of array
    if ((bx*BlockSize+tx)>=Nfits) return;
    
	memset(M,0,NV_P*NV_P*sizeof(float));
	memset(Minv,0,NV_P*NV_P*sizeof(float));
    //load data
    const float *s_data = d_data+(sz*sz*bx*BlockSize+sz*sz*tx);
	const float *s_Coords = d_Coords+(2*bx*BlockSize+2*tx);
	
    //initial values
    kernel_CenterofMass2D(sz, s_data, &theta[0], &theta[1]);
    kernel_GaussFMaxMin2D(sz, PSFSigma, s_data, &Nmax, &theta[3]);
    theta[2]=max(0.0f, (Nmax-theta[3])*2*2*pi*PSFSigma*PSFSigma);
    
    for (kk=0;kk<iterations;kk++) {//main iterative loop
        
        //initialize
		memset(NR_Numerator,0,NV_P*sizeof(float));
		memset(NR_Denominator,0,NV_P*sizeof(float));

        for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
            PSFx=kernel_IntGauss1D(ii, theta[0], PSFSigma);
            PSFy=kernel_IntGauss1D(jj, theta[1], PSFSigma);
            
            model=theta[3]+theta[2]*PSFx*PSFy;
            data=s_data[sz*jj+ii];
            GRind=(s_Coords[1]+jj)*Mapsz+s_Coords[0]+ii;
			//GRind=(int)s_Coords[0];
			gainR=d_GainRatio[GRind];
            //calculating derivatives
            kernel_DerivativeIntGauss1D(ii, theta[0], PSFSigma, theta[2], PSFy, &dudt[0], &d2udt2[0]);//x
            kernel_DerivativeIntGauss1D(jj, theta[1], PSFSigma, theta[2], PSFx, &dudt[1], &d2udt2[1]);//y
            dudt[2] = PSFx*PSFy;// I
            d2udt2[2] = 0.0f;// I
            dudt[3] = 1.0f;// bg
            d2udt2[3] = 0.0f;// bg
            
            cf=0.0f;
            df=0.0f;
            if (model>10e-3f) cf=(data-model)/(model+gainR); // add variance-gain ratio: v/g^2
            if (model>10e-3f) df=(data+gainR)/pow(model+gainR, 2); // add variance-gain ratio: v/g^2
            cf=min(cf, 10e4f);
            df=min(df, 10e4f);
            
            for (ll=0;ll<NV;ll++){
                NR_Numerator[ll]+=dudt[ll]*cf;
                NR_Denominator[ll]+=d2udt2[ll]*cf-pow(dudt[ll], 2)*df;
            }
        }
        
        // The update: Editted 8.9.16 to allow for larger jumps in Photons/BG
        theta[0]-=min(max(NR_Numerator[0]/NR_Denominator[0], -maxjump[0]), maxjump[0]);
        theta[1]-=min(max(NR_Numerator[1]/NR_Denominator[1], -maxjump[1]), maxjump[1]);
        theta[2]-=min(max(NR_Numerator[2]/NR_Denominator[2], -theta[2]), theta[2]);
        theta[3]-=min(max(NR_Numerator[3]/NR_Denominator[3], -theta[3]), theta[3]);

        // Any other constraints
        theta[2]=max(theta[2], 1.0f); //Make sure Photons is postitve
        theta[3]=max(theta[3], 0.01f); //Make sure Background is postitve
        
    }
    
    // Calculating the CRLB and LogLikelihood
    Div=0.0;
    for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
        PSFx=kernel_IntGauss1D(ii, theta[0], PSFSigma);
        PSFy=kernel_IntGauss1D(jj, theta[1], PSFSigma);
        
        model=theta[3]+theta[2]*PSFx*PSFy;
        data=s_data[sz*jj+ii];
        
        //calculating derivatives
        kernel_DerivativeIntGauss1D(ii, theta[0], PSFSigma, theta[2], PSFy, &dudt[0], NULL);
        kernel_DerivativeIntGauss1D(jj, theta[1], PSFSigma, theta[2], PSFx, &dudt[1], NULL);
        dudt[2] = PSFx*PSFy;
        dudt[3] = 1.0f;
        
        //Building the Fisher Information Matrix
        for (kk=0;kk<NV;kk++)for (ll=kk;ll<NV;ll++){
            M[kk*NV+ll]+= dudt[ll]*dudt[kk]/(model+gainR);// add gain ratio
            M[ll*NV+kk]=M[kk*NV+ll];
        }
        
        //LogLikelyhood
        if ((model+gainR)>0)
            if ((data+gainR)>0)Div+=(data+gainR)*log(model+gainR)-model-(data+gainR)*log(data+gainR)+data;// add gain ratio
            else
                Div+=-model-gainR;
    }
    
    // Matrix inverse (CRLB=F^-1) and output assigments
    kernel_MatInvN(M, Minv, Diag, NV);
    
    //write to global arrays
    for (kk=0;kk<NV;kk++) d_Parameters[Nfits*kk+BlockSize*bx+tx]=theta[kk];
    for (kk=0;kk<NV;kk++) d_CRLBs[Nfits*kk+BlockSize*bx+tx]=Diag[kk];
    d_LogLikelihood[BlockSize*bx+tx] = Div;
    
    return;
}


//*******************************************************************************************
__global__ void kernel_MLEFit_SCMOSXYNBS_(const float *d_data, const float *d_Coords, const float *d_GainRatio,
	    const float PSFSigma, const int sz, const int Mapsz, const int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,const int Nfits){
	/*! 
	 * \brief basic MLE fitting kernel.  No additional parameters are computed.
	 * \param d_data array of subregions to fit copied to GPU
	 * \param d_Coords array of subregions's pixel coordinates in original field of view.
	 * \param d_GainRatio calibration result of variance/gain^2 in each pixel of original field of view.
	 * \param PSFSigma sigma of the point spread function
	 * \param sz nxn size of the subregion to fit
	 * \param Mapsz size of original field of view.
	 * \param iterations number of iterations for solution to converge
	 * \param d_Parameters array of fitting parameters to return for each subregion
	 * \param d_CRLBs array of Cramer-Rao lower bound estimates to return for each subregion
	 * \param d_LogLikelihood array of loglikelihood estimates to return for each subregion
	 * \param Nfits number of subregions to fit
	 */
    
    //__shared__ float s_data[MEM];
    float M[NV_PS*NV_PS], Diag[NV_PS], Minv[NV_PS*NV_PS];
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int BlockSize = blockDim.x;
    int ii, jj, kk, ll;
    float model, cf, df, data;
    float Div;
    float PSFy, PSFx;
    int NV=NV_PS;
    float dudt[NV_PS];
    float d2udt2[NV_PS];
    float NR_Numerator[NV_PS], NR_Denominator[NV_PS];
    float theta[NV_PS];
    float maxjump[NV_PS]={1e0f, 1e0f, 1e2f, 2e0f, 5e-1f};
    float Nmax;
    float gainR;
	int GRind;
    //Prevent read/write past end of array
    if ((bx*BlockSize+tx)>=Nfits) return;
    
	memset(M,0,NV_PS*NV_PS*sizeof(float));
	memset(Minv,0,NV_PS*NV_PS*sizeof(float));      
    //load data
    const float *s_data = d_data+(sz*sz*bx*BlockSize+sz*sz*tx);
    const float *s_Coords = d_Coords+(2*bx*BlockSize+2*tx);
    //initial values
    kernel_CenterofMass2D(sz, s_data, &theta[0], &theta[1]);
    kernel_GaussFMaxMin2D(sz, PSFSigma, s_data, &Nmax, &theta[3]);
    theta[2]=max(0.0f, (Nmax-theta[3])*2*2*pi*PSFSigma*PSFSigma);
    theta[4]=PSFSigma;
    
    for (kk=0;kk<iterations;kk++) {//main iterative loop
        
        //initialize
  		memset(NR_Numerator,0,NV_PS*sizeof(float));
		memset(NR_Denominator,0,NV_PS*sizeof(float));
      
        for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
            PSFx=kernel_IntGauss1D(ii, theta[0], theta[4]);
            PSFy=kernel_IntGauss1D(jj, theta[1], theta[4]);
            
            model=theta[3]+theta[2]*PSFx*PSFy;
            data=s_data[sz*jj+ii];
            GRind=(s_Coords[1]+jj)*Mapsz+s_Coords[0]+ii;
			gainR=d_GainRatio[GRind];
            //calculating derivatives
            kernel_DerivativeIntGauss1D(ii, theta[0], theta[4], theta[2], PSFy, &dudt[0], &d2udt2[0]);
            kernel_DerivativeIntGauss1D(jj, theta[1], theta[4], theta[2], PSFx, &dudt[1], &d2udt2[1]);
            kernel_DerivativeIntGauss2DSigma(ii, jj, theta[0], theta[1], theta[4], theta[2], PSFx, PSFy, &dudt[4], &d2udt2[4]);
            dudt[2] = PSFx*PSFy;
            d2udt2[2] = 0.0f;
            dudt[3] = 1.0f;
            d2udt2[3] = 0.0f;
            
            cf=0.0f;
            df=0.0f;
            if (model>10e-3f) cf=(data-model)/(model+gainR);
            if (model>10e-3f) df=(data+gainR)/pow(model+gainR, 2);
            cf=min(cf, 10e4f);
            df=min(df, 10e4f);
            
            for (ll=0;ll<NV;ll++){
                NR_Numerator[ll]+=dudt[ll]*cf;
                NR_Denominator[ll]+=d2udt2[ll]*cf-pow(dudt[ll], 2)*df;
            }
        }
        
        // The update: Editted 8.9.16 to allow for larger jumps in Photons/BG
        theta[0]-=min(max(NR_Numerator[0]/NR_Denominator[0], -maxjump[0]), maxjump[0]);
        theta[1]-=min(max(NR_Numerator[1]/NR_Denominator[1], -maxjump[1]), maxjump[1]);
        theta[2]-=min(max(NR_Numerator[2]/NR_Denominator[2], -theta[2]), theta[2]);
        theta[3]-=min(max(NR_Numerator[3]/NR_Denominator[3], -theta[3]), theta[3]);
        theta[4]-=min(max(NR_Numerator[4]/NR_Denominator[4], -theta[4]), theta[4]);

        // Any other constraints
        theta[2]=max(theta[2], 1.0f); //Make sure Photons is postitve
        theta[3]=max(theta[3], 0.01f); //Make sure Background is postitve
        theta[4]=max(theta[4], 0.5f); //Constrain Sigma
        theta[4]=min(theta[4], sz/2.0f); //Constrain Sigma
    }
    
    // Calculating the CRLB and LogLikelihood
    Div=0.0f;
    for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
        PSFx=kernel_IntGauss1D(ii, theta[0], theta[4]);
        PSFy=kernel_IntGauss1D(jj, theta[1], theta[4]);
        
        model=theta[3]+theta[2]*PSFx*PSFy;
        data=s_data[sz*jj+ii];
        
        //calculating derivatives
        kernel_DerivativeIntGauss1D(ii, theta[0], theta[4], theta[2], PSFy, &dudt[0], NULL);
        kernel_DerivativeIntGauss1D(jj, theta[1], theta[4], theta[2], PSFx, &dudt[1], NULL);
        kernel_DerivativeIntGauss2DSigma(ii, jj, theta[0], theta[1], theta[4], theta[2], PSFx, PSFy, &dudt[4], NULL);
        dudt[2] = PSFx*PSFy;
        dudt[3] = 1.0f;
        
        //Building the Fisher Information Matrix
        for (kk=0;kk<NV;kk++)for (ll=kk;ll<NV;ll++){
            M[kk*NV+ll]+= dudt[ll]*dudt[kk]/(model+gainR);
            M[ll*NV+kk]=M[kk*NV+ll];
        }
        
        //LogLikelyhood
        if ((model+gainR)>0)
            if ((data+gainR)>0)Div+=(data+gainR)*log(model+gainR)-model-(data+gainR)*log(data+gainR)+data;
            else
                Div+=-model-gainR;
    }
    
    // Matrix inverse (CRLB=F^-1) and output assigments
    kernel_MatInvN(M, Minv, Diag, NV);
  
    
    //write to global arrays
    for (kk=0;kk<NV;kk++) d_Parameters[Nfits*kk+BlockSize*bx+tx]=theta[kk];
    for (kk=0;kk<NV;kk++) d_CRLBs[Nfits*kk+BlockSize*bx+tx]=Diag[kk];
    d_LogLikelihood[BlockSize*bx+tx] = Div;
    
    return;
}

//*******************************************************************************************
__global__ void kernel_MLEFit_SCMOSXYNBZ_(const float *d_data, const float *d_Coords, const float *d_GainRatio, const float *d_x0,
		const float PSFSigma_x, const float Ax, const float Ay, const float Bx, 
		const float By, const float gamma, const float d, const float PSFSigma_y, const int sz, const int Mapsz, const int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,const int Nfits){
	/*! 
	 * \brief basic MLE fitting kernel.  No additional parameters are computed.
	 * \param d_data array of subregions to fit copied to GPU
	 * \param d_Coords array of subregions's pixel coordinates in original field of view.
	 * \param d_GainRatio calibration result of variance/gain^2 in each pixel of original field of view. 
	 * \param PSFSigma_x sigma of the point spread function on the x axis
	 * \param Ax ???
	 * \param Ay ???
	 * \param Bx ???
	 * \param By ???
	 * \param gamma ???
	 * \param d ???
	 * \param PSFSigma_y sigma of the point spread function on the y axis
	 * \param sz nxn size of the subregion to fit
	 * \param Mapsz size of original field of view.
	 * \param iterations number of iterations for solution to converge
	 * \param d_Parameters array of fitting parameters to return for each subregion
	 * \param d_CRLBs array of Cramer-Rao lower bound estimates to return for each subregion
	 * \param d_LogLikelihood array of loglikelihood estimates to return for each subregion
	 * \param Nfits number of subregions to fit
	 */
    //__shared__ float s_data[MEM];
    float M[5*5], Diag[5], Minv[5*5];
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int BlockSize = blockDim.x;
    int ii, jj, kk, ll;
    float model, cf, df, data;
    float Div;
    float PSFy, PSFx;
    int NV=5;
    float dudt[5];
    float d2udt2[5];
    float NR_Numerator[5], NR_Denominator[5];
    float theta[5];
    float maxjump[5]={1e0f, 1e0f, 1e2f, 2e0f, 1e-1f};
    float Nmax;
    float gainR;
	int GRind;
    //Prevent read/write past end of array
    if ((bx*BlockSize+tx)>=Nfits) return;

	memset(M,0,NV*NV*sizeof(float));
	memset(Minv,0,NV*NV*sizeof(float));      
    
    //load data
    const float *s_data = d_data+(sz*sz*bx*BlockSize+sz*sz*tx);
	const float *s_Coords = d_Coords+(2*bx*BlockSize+2*tx);
	const float *z_initial = d_x0+(bx*BlockSize+tx);
    //initial values
    kernel_CenterofMass2D(sz, s_data, &theta[0], &theta[1]);
    kernel_GaussFMaxMin2D(sz, PSFSigma_x, s_data, &Nmax, &theta[3]);
    theta[2]=max(0.0f, (Nmax-theta[3])*2*pi*PSFSigma_x*PSFSigma_y*sqrt(2.0f));
    theta[4]=z_initial[0];
   
    for (kk=0;kk<iterations;kk++) {//main iterative loop
        
        //initialize
  		memset(NR_Numerator,0,NV*sizeof(float));
		memset(NR_Denominator,0,NV*sizeof(float));
        
        for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
            kernel_DerivativeIntGauss2Dz(ii, jj, theta, PSFSigma_x,PSFSigma_y, Ax,Ay,Bx,By, gamma, d, &PSFx, &PSFy, dudt, d2udt2);
            
            model=theta[3]+theta[2]*PSFx*PSFy;
            data=s_data[sz*jj+ii];
            GRind=(s_Coords[1]+jj)*Mapsz+s_Coords[0]+ii;
			gainR=d_GainRatio[GRind];
            //calculating remaining derivatives
            dudt[2] = PSFx*PSFy;
            d2udt2[2] = 0.0f;
            dudt[3] = 1.0f;
            d2udt2[3] = 0.0f;
            
            cf=0.0f;
            df=0.0f;
            if (model>10e-3f) cf=(data-model)/(model+gainR);
            if (model>10e-3f) df=(data+gainR)/pow(model+gainR, 2);
            cf=min(cf, 10e4f);
            df=min(df, 10e4f);
            
            for (ll=0;ll<NV;ll++){
                NR_Numerator[ll]+=dudt[ll]*cf;
                NR_Denominator[ll]+=d2udt2[ll]*cf-pow(dudt[ll], 2)*df;
            }
        }
        
         // The update: Editted 8.9.16 to allow for larger jumps in Photons/BG
        theta[0]-=min(max(NR_Numerator[0]/NR_Denominator[0], -maxjump[0]), maxjump[0]);
        theta[1]-=min(max(NR_Numerator[1]/NR_Denominator[1], -maxjump[1]), maxjump[1]);
        theta[2]-=min(max(NR_Numerator[2]/NR_Denominator[2], -theta[2]), theta[2]);
        theta[3]-=min(max(NR_Numerator[3]/NR_Denominator[3], -theta[3]), theta[3]);
        theta[4]-=min(max(NR_Numerator[4]/NR_Denominator[4], -maxjump[4]), maxjump[4]);
        
        // Any other constraints
        theta[2]=max(theta[2], 1.0f);
        theta[3]=max(theta[3], 0.01f);
        
    }
    
    // Calculating the CRLB and LogLikelihood
    Div=0.0f;
    for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
        
        kernel_DerivativeIntGauss2Dz(ii, jj, theta, PSFSigma_x,PSFSigma_y, Ax,Ay, Bx,By, gamma, d, &PSFx, &PSFy, dudt, NULL);
        
        model=theta[3]+theta[2]*PSFx*PSFy;
        data=s_data[sz*jj+ii];
        
        //calculating remaining derivatives
        dudt[2] = PSFx*PSFy;
        dudt[3] = 1.0f;
       
        //Building the Fisher Information Matrix
        for (kk=0;kk<NV;kk++)for (ll=kk;ll<NV;ll++){
            M[kk*NV+ll]+= dudt[ll]*dudt[kk]/model;
            M[ll*NV+kk]=M[kk*NV+ll];
        }
        
        //LogLikelyhood
        if ((model+gainR)>0)
            if ((data+gainR)>0)Div+=(data+gainR)*log(model+gainR)-model-(data+gainR)*log(data+gainR)+data;
            else
                Div+=-model-gainR;
    }
    
    // Matrix inverse (CRLB=F^-1) 
    kernel_MatInvN(M, Minv, Diag, NV);
  
   //write to global arrays
    for (kk=0;kk<NV;kk++) d_Parameters[Nfits*kk+BlockSize*bx+tx]=theta[kk];
    for (kk=0;kk<NV;kk++) d_CRLBs[Nfits*kk+BlockSize*bx+tx]=Diag[kk];
    d_LogLikelihood[BlockSize*bx+tx] = Div;
    return;
}

//*******************************************************************************************
__global__ void kernel_MLEFit_SCMOSXYNBSXSY_(const float *d_data, const float *d_Coords, const float *d_GainRatio, 
	    const float PSFSigma, const int sz, const int Mapsz, const int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,const int Nfits){
	/*! 
	 * \brief basic MLE fitting kernel.  No additional parameters are computed.
	 * \param d_data array of subregions to fit copied to GPU
	 * \param d_Coords array of subregions's pixel coordinates in original field of view.
	 * \param d_GainRatio calibration result of variance/gain^2 in each pixel of original field of view. 
	 * \param PSFSigma sigma of the point spread function
	 * \param sz nxn size of the subregion to fit
	 * \param Mapsz size of original field of view.
	 * \param iterations number of iterations for solution to converge
	 * \param d_Parameters array of fitting parameters to return for each subregion
	 * \param d_CRLBs array of Cramer-Rao lower bound estimates to return for each subregion
	 * \param d_LogLikelihood array of loglikelihood estimates to return for each subregion
	 * \param Nfits number of subregions to fit
	 */
 
    //__shared__ float s_data[MEM];
    float M[6*6], Diag[6], Minv[6*6];
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int BlockSize = blockDim.x;
    int ii, jj, kk, ll;
    float model, cf, df, data;
    float Div;
    float PSFy, PSFx;
    int NV=6;
    float dudt[6];
    float d2udt2[6];
    float NR_Numerator[6], NR_Denominator[6];
    float theta[6];
    float maxjump[6]={1e0f, 1e0f, 1e2f, 2e0f, 1e-1f,1e-1f};
    float Nmax;
    float gainR;
	int GRind;
    //Prevent read/write past end of array
    if ((bx*BlockSize+tx)>=Nfits) return;
    
	memset(M,0,NV*NV*sizeof(float));
	memset(Minv,0,NV*NV*sizeof(float));      
    
    //load data
    const float *s_data = d_data+(sz*sz*bx*BlockSize+sz*sz*tx);
    const float *s_Coords = d_Coords+(2*bx*BlockSize+2*tx);
	//initial values
    kernel_CenterofMass2D(sz, s_data, &theta[0], &theta[1]);
    kernel_GaussFMaxMin2D(sz, PSFSigma, s_data, &Nmax, &theta[3]);
    theta[2]=max(0.0f, (Nmax-theta[3])*2*2*pi*PSFSigma*PSFSigma);
    theta[4]=PSFSigma;
    theta[5]=PSFSigma;
    for (kk=0;kk<iterations;kk++) {//main iterative loop
        
        //initialize
  		memset(NR_Numerator,0,NV*sizeof(float));
		memset(NR_Denominator,0,NV*sizeof(float));
        
        for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
            PSFx=kernel_IntGauss1D(ii, theta[0], theta[4]);
            PSFy=kernel_IntGauss1D(jj, theta[1], theta[5]);
            
            model=theta[3]+theta[2]*PSFx*PSFy;
            data=s_data[sz*jj+ii];
            GRind=(s_Coords[1]+jj)*Mapsz+s_Coords[0]+ii;
			gainR=d_GainRatio[GRind];
            //calculating derivatives
            kernel_DerivativeIntGauss1D(ii, theta[0], theta[4], theta[2], PSFy, &dudt[0], &d2udt2[0]);
            kernel_DerivativeIntGauss1D(jj, theta[1], theta[5], theta[2], PSFx, &dudt[1], &d2udt2[1]);
            kernel_DerivativeIntGauss1DSigma(ii, theta[0], theta[4], theta[2], PSFy, &dudt[4], &d2udt2[4]);
            kernel_DerivativeIntGauss1DSigma(jj, theta[1], theta[5], theta[2], PSFx, &dudt[5], &d2udt2[5]);
            dudt[2] = PSFx*PSFy;
            d2udt2[2] = 0.0f;
            dudt[3] = 1.0f;
            d2udt2[3] = 0.0f;
            
            cf=0.0f;
            df=0.0f;
            if (model>10e-3f) cf=(data-model)/(model+gainR);
            if (model>10e-3f) df=(data+gainR)/pow(model+gainR, 2);
            cf=min(cf, 10e4f);
            df=min(df, 10e4f);
            
            for (ll=0;ll<NV;ll++){
                NR_Numerator[ll]+=dudt[ll]*cf;
                NR_Denominator[ll]+=d2udt2[ll]*cf-pow(dudt[ll], 2)*df;
            }
        }
        
         // The update: Editted 8.9.16 to allow for larger jumps in Photons/BG
        theta[0]-=min(max(NR_Numerator[0]/NR_Denominator[0], -maxjump[0]), maxjump[0]);
        theta[1]-=min(max(NR_Numerator[1]/NR_Denominator[1], -maxjump[1]), maxjump[1]);
        theta[2]-=min(max(NR_Numerator[2]/NR_Denominator[2], -theta[2]), theta[2]);
        theta[3]-=min(max(NR_Numerator[3]/NR_Denominator[3], -theta[3]), theta[3]);
        theta[4]-=min(max(NR_Numerator[4]/NR_Denominator[4], -theta[4]), theta[4]);
        theta[5]-=min(max(NR_Numerator[5]/NR_Denominator[5], -theta[5]), theta[5]);

        // Any other constraints
        theta[2]=max(theta[2], 1.0f); //Make sure Photons is postitve
        theta[3]=max(theta[3], 0.01f); //Make sure Background is postitve
        theta[4]=max(theta[4], 0.5f); //Constrain Sigma
        theta[4]=min(theta[4], sz/2.0f); //Constrain SigmaX
        theta[5]=max(theta[5], 0.5f); //Constrain Sigma
        theta[5]=min(theta[5], sz/2.0f); //Constrain SigmaX
    }
    
    // Calculating the CRLB and LogLikelihood
    Div=0.0f;
    for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
        
        PSFx=kernel_IntGauss1D(ii, theta[0], theta[4]);
        PSFy=kernel_IntGauss1D(jj, theta[1], theta[5]);
        
        model=theta[3]+theta[2]*PSFx*PSFy;
        data=s_data[sz*jj+ii];
        
        //calculating derivatives
        kernel_DerivativeIntGauss1D(ii, theta[0], theta[4], theta[2], PSFy, &dudt[0], NULL);
        kernel_DerivativeIntGauss1D(jj, theta[1], theta[5], theta[2], PSFx, &dudt[1], NULL);
        kernel_DerivativeIntGauss1DSigma(ii, theta[0], theta[4], theta[2], PSFy, &dudt[4], NULL);
        kernel_DerivativeIntGauss1DSigma(jj, theta[1], theta[5], theta[2], PSFx, &dudt[5], NULL);
        dudt[2] = PSFx*PSFy;
        dudt[3] = 1.0f;
        
        //Building the Fisher Information Matrix
        for (kk=0;kk<NV;kk++)for (ll=kk;ll<NV;ll++){
            M[kk*NV+ll]+= dudt[ll]*dudt[kk]/model;
            M[ll*NV+kk]=M[kk*NV+ll];
        }
        
        //LogLikelyhood
        if ((model+gainR)>0)
            if ((data+gainR)>0)Div+=(data+gainR)*log(model+gainR)-model-(data+gainR)*log(data+gainR)+data;
            else
                Div+=-model-gainR;
    }
    
    // Matrix inverse (CRLB=F^-1) and output assigments
    kernel_MatInvN(M, Minv, Diag, NV);
   
    //write to global arrays
    for (kk=0;kk<NV;kk++) d_Parameters[Nfits*kk+BlockSize*bx+tx]=theta[kk];
    for (kk=0;kk<NV;kk++) d_CRLBs[Nfits*kk+BlockSize*bx+tx]=Diag[kk];
    d_LogLikelihood[BlockSize*bx+tx] = Div;
    return;
}

#define NUM_FIT 8
#define a1 -0.9911f
#define a2 -0.6763f
#define b1 0.8055f
#define b2 -1.2451f
	//********************************************************************************************************************************************
__global__ void kernel_gaussMFA(const float *d_data, const float PSFSigma, 
        const int sz, const int iterations,const int num, const float Nave, 
                 const float llThreshold, float *d_X, 
                        float *d_Y, float *d_b, float *d_PValue, int Nfits) 
{
	
	float Nmax;

	int tx = threadIdx.x; //fits
	int bx = blockIdx.x;
    int BlockSize = blockDim.x;

	int ii, jj, kk, qq, mm, nn, nnM, goodness;
	int nnfit=0;
	int breaktmp, nncount, rimsigny, rimsignx;
	float dLLx, dLLy, dLLb;
	float deflamodel;
	float tmp, signx, signy, meanx, meany;
	float b, x, y, xmin, xmax, ymin, ymax;
	float model, cf, df, data,bini;
	float Div;
	float bfit=0;
	float imdy, imdx,  imddx, imddy;
	float PSFy, PSFx, pval, zval,minDiv,maxpval;
	float stepxtot;
	float stepytot;
	float stepbgtot;
	float xarray[NUM_FIT]={0, 0, 0, 0, 0, 0, 0, 0};
	float yarray[NUM_FIT]={0, 0, 0, 0, 0, 0, 0, 0};
	float xarrayfit[NUM_FIT]={0, 0, 0, 0, 0, 0, 0, 0};
	float yarrayfit[NUM_FIT]={0, 0, 0, 0, 0, 0, 0, 0};
	float x_current[NUM_FIT]={0, 0, 0, 0, 0, 0, 0, 0};
	float y_current[NUM_FIT]={0, 0, 0, 0, 0, 0, 0, 0};

	//load data
    const float *s_data = d_data+(sz*sz*bx*BlockSize+sz*sz*tx);

	//Prevent read/write past end of array
    if ((bx*BlockSize+tx)>=Nfits) return;

	//initial values
	kernel_CenterofMass2D(sz, s_data, &x, &y);
	kernel_GaussFMaxMin2D(sz, PSFSigma, s_data, &Nmax, &b);
	bini=max( b, 0.0001f);

	PSFy = 0.0f; PSFx=0.0f;

	maxpval=-1.0f; //this keeps track of models' pvalue
	
    //loop over number of emitters
	for (mm=1;mm<=num;mm++) {
		breaktmp=0;
		goodness=0;
		b=bini;

		//INITIAL GUESSES

		//if fitting for a single fluorophore, use center of mass as initial guess
		if (mm==1){
			xarray[0]=x;
			yarray[0]=y;
		}

		//if fitting more than one fluorophore, use deflation method and find max
		else {
			tmp=Nave/2/pi/PSFSigma/PSFSigma/10;

			for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
				model=b;
				nncount=0;
				meanx=0;
				meany=0;
				for (nn=0;nn<mm-1;nn++) {
					x=xarray[nn];
					y=yarray[nn];
					meanx+=xarray[nn];
					meany+=yarray[nn];
					nncount=nncount+1;
					PSFx=kernel_IntGauss1D(ii, x, PSFSigma);
					PSFy=kernel_IntGauss1D(jj, y, PSFSigma);					
					model+=Nave*PSFx*PSFy;
				}
				
				deflamodel=s_data[sz*jj+ii]-model;
				signx=0;
				signy=0;
				rimsignx=1;
				rimsigny=1;
				if (deflamodel>tmp) {
					if (ii==0) rimsignx=-1;
					if (ii==sz-1) rimsignx=-1;
					if (jj==0) rimsigny=-1;
					if (jj==sz-1) rimsigny=-1;

					signx=((ii-meanx/nncount) > 0)? 1 : -1;
					signy=((jj-meany/nncount) > 0)? 1 : -1;
					xarray[mm-1]=ii-0.5f*signx*rimsignx;
					yarray[mm-1]=jj-0.5f*signy*rimsigny;
					tmp=deflamodel;
				}
			}
			if (tmp==Nave/2/pi/PSFSigma/PSFSigma/10)
				breaktmp=1;
		}

		if (breaktmp==1)
			break;

		//MAIN ITERATIVE LOOP
		for (kk=0;kk<iterations;kk++){

			dLLb=0.0f;
			stepbgtot = 0;

			//generating the model and calc b update
			for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {

				model=b;
				for (nn=0;nn<mm;nn++) {
					x_current[nn]=xarray[nn];
					y_current[nn]=yarray[nn];

					x=xarray[nn];
					y=yarray[nn];

					PSFx=kernel_IntGauss1D(ii, x, PSFSigma);
					PSFy=kernel_IntGauss1D(jj, y, PSFSigma);
					model+=Nave*PSFx*PSFy;
				}
				data=s_data[sz*jj+ii];
				cf=0.0f;
				df=0.0f;
				if (model>10e-3f) cf=data/model-1;
				if (model>10e-3f) df=data/pow(model, 2);
				cf=min( cf, 10e4f);
				df=min( df, 10e4f);
				stepbgtot += -df;
				dLLb+=cf;
			}
			b-=min( max( dLLb/stepbgtot, -1e0f), 1e0f)/mm/2;
			b=max( b, 0.001f);

			//This starts iterative routine for theta_i other than theta_bg which is calculated above.
			for (nn=0;nn<mm;nn++){

				dLLx=0.0f;
				dLLy=0.0f;
				stepxtot = 0.0f;
				stepytot = 0.0f;

				for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
					//generate model using initial value or iteration value.
					data=s_data[sz*jj+ii];
					//data=s_data[sz*sz*tx+sz*jj+ii];
					model=b;
					for (nnM=0;nnM<mm;nnM++) {
						x=x_current[nnM];
						y=y_current[nnM];
						PSFx=kernel_IntGauss1D(ii, x, PSFSigma);
						PSFy=kernel_IntGauss1D(jj, y, PSFSigma);
						model+=Nave*PSFx*PSFy;
					}

					cf=0.0f;
					df=0.0f;
					if (model>10e-3f) cf=data/model-1;
					if (model>10e-3f) df=data/pow( model, 2);
					cf=min( cf, 10e4f);
					df=min( df, 10e4f);

					x=x_current[nn];
					y=y_current[nn];

					PSFx=kernel_IntGauss1D(ii, x, PSFSigma);
					PSFy=kernel_IntGauss1D(jj, y, PSFSigma);
					
					kernel_DerivativeIntGauss1D(ii, x, PSFSigma, Nave, PSFy, &imdx, &imddx);
					kernel_DerivativeIntGauss1D(jj, y, PSFSigma, Nave, PSFx, &imdy, &imddy);

					//denominator
					stepxtot  += imddx*cf-pow( imdx, 2)*df;
					stepytot  += imddy*cf-pow( imdy, 2)*df;

					//numerator
					dLLx+=imdx*cf;
					dLLy+=imdy*cf;
				}

				x-=min( max( dLLx/stepxtot, -1e0f), 1e0f)/mm/2.0f;
				y-=min( max( dLLy/stepytot, -1e-0f), 1e-0f)/mm/2.0f;
				xarray[nn]=x;
				yarray[nn]=y;
			}
		}

		// calculating loglikelihood value
		Div=0.0f;
		xmin=1000;
		xmax=-1000;
		ymin=1000;
		ymax=-1000;

		for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
			// generating the model
			model=b;
			for (nn=0;nn<mm;nn++) {
				x=xarray[nn];
				y=yarray[nn];
                
                //reject this fit model if too far outside
				if (x>xmax) xmax=x;
				if (x<xmin) xmin=x;
				if (y>ymax) ymax=y;
				if (y<ymin) ymin=y;

				PSFx=kernel_IntGauss1D(ii, x, PSFSigma);
				PSFy=kernel_IntGauss1D(jj, y, PSFSigma);
				model+=Nave*PSFx*PSFy;
			}

			data=s_data[sz*jj+ii];

			if (data>0){
				Div+=-2*(data*log(model)-model-data*log(data)+data-0.5f*log(2*pi*data)-model*log(model)+model+model*log(model)-model+0.5f*log(2*pi*model));}

			else{
				Div+=-2*(-model-model*log(model)+model+model*log(model)-model+0.5f*log(2*pi*model));}
		}

		zval=sqrt((float) Div)-sqrt((float) (sz*sz-2*mm-1));
		pval=(zval<0)*(1-0.5f*exp(b1*zval+a1*pow(zval,2)))+(zval>0)*(0.5f*exp(b2*zval+a2*pow(zval,2)));

        //if this is true, model is better than last
		if ((pval>maxpval) && (xmin>-round(1.5f*PSFSigma)) && (xmax <(sz-1+round(1.5f*PSFSigma))) && (ymin>-round(1.5f*PSFSigma)) && (ymax<(sz-1+round(1.5f*PSFSigma)))) 
        {
			for (qq=0;qq<mm;qq++)
			{
				xarrayfit[qq]=xarray[qq];
				yarrayfit[qq]=yarray[qq];
			}
			maxpval=pval;
			minDiv=Div;
			bfit=b;
			nnfit=mm;
		}
	}

	if ((xarrayfit[0]!=0)&&(maxpval>llThreshold*0.01)){
		b=bfit;
		for (qq=0;qq<nnfit;qq++)
		{
			xarray[qq]=xarrayfit[qq];
			yarray[qq]=yarrayfit[qq];
		}	
	}

	// output
	if (maxpval>llThreshold)
		goodness=1;

	int fitnum=num;

	for (nn=0;nn<fitnum;nn++)
		d_X[BlockSize*fitnum*bx+fitnum*tx+nn]=xarrayfit[nn]*goodness;

	for (nn=0;nn<fitnum;nn++)
		d_Y[BlockSize*fitnum*bx+fitnum*tx+nn]=yarrayfit[nn]*goodness;

	d_b[BlockSize*bx+tx]=bfit*goodness;
	d_PValue[BlockSize*bx+tx]=maxpval;
	return;
}
#define NMAX 5
#define MEM_CRLB (2*NMAX+1)
#define MATMEM (2*NMAX+1)*(2*NMAX+1)

__global__ void kernel_CRLB(const int sz, const int fitnum,const float Nave, 
							const float PSFSigma, const float *d_xarray, const float *d_yarray, 
							const float *d_barray, float *d_CRLBarray, 
							float *d_covariance, const int Nfits,  float *d_crlbx, float *d_crlby) 
{

	int tx = threadIdx.x; //fits
	int bx = blockIdx.x;
	int BlockSize = blockDim.x;

	int kk, ll, ii, jj,nn;
	float x, y;
	float PSFx, PSFy, imdx,imdy;
	float imddx, imddy;
	float imdbg;
	int fitstate, matsz;
	int matsize=0,numco;

	float model,disx,disy;//, *s_Diag=0, *s_Diag2=0;
	float s_Diag[MEM_CRLB]; //x,y,bg
	float s_temp[MATMEM];
	float s_fishermatrix[MATMEM];
	float zerofisher[MATMEM];

	for (kk=0;kk<MATMEM;kk++){
		s_fishermatrix[kk]=0;
		s_temp[kk]=0;
	}

	// data array locations
	const float *s_xarray = d_xarray+(fitnum*bx*BlockSize+fitnum*tx);
	const float *s_yarray = d_yarray+(fitnum*bx*BlockSize+fitnum*tx);
	const float *s_barray = d_barray+(bx*BlockSize+tx);

	// adding variables for CRLB_x and CRLB_y
	float *s_crlbx = d_crlbx+(fitnum*bx*BlockSize+fitnum*tx);
	float *s_crlby = d_crlby+(fitnum*bx*BlockSize+fitnum*tx);

	//Prevent read/write past end of array
    if ((bx*BlockSize+tx)>=Nfits) return;

	//calculation starts.

	for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
		// generate model for pixel ii jj

		model=s_barray[tx];

		for (nn=0;nn<fitnum;nn++){
			fitstate=0;
			x=s_xarray[nn];
			y=s_yarray[nn];
			if(x!=0) fitstate=1;
			PSFx=kernel_IntGauss1D(ii, x, PSFSigma)*fitstate;
			PSFy=kernel_IntGauss1D(jj, y, PSFSigma)*fitstate;
			model+=Nave*PSFx*PSFy;
		}

		matsize=0;
		for (nn=0;nn<fitnum;nn++){
			fitstate=0;
			x=s_xarray[nn];
			y=s_yarray[nn];
			if(x!=0) fitstate=1;
			PSFx=kernel_IntGauss1D(ii, x, PSFSigma)*fitstate;
			PSFy=kernel_IntGauss1D(jj, y, PSFSigma)*fitstate;
			
			kernel_DerivativeIntGauss1D(ii, x, PSFSigma, Nave, PSFy, &imdx, &imddx );
			kernel_DerivativeIntGauss1D(jj, y, PSFSigma, Nave, PSFx, &imdy, &imddy );
			imdx = imdx*fitstate;
			imdy = imdy*fitstate;

			//store in temp memory (which's allocate for LUDC method)
			s_temp[2*nn]=imdx;
			s_temp[2*nn+1]=imdy;
			matsize+=fitstate;
		}
		imdbg = 1.0f;
		// put derivitive of bg right after fitted parameters
		if (matsize>0)
			s_temp[2*matsize]=imdbg;

		for (kk=0;kk<(2*matsize+1);kk++)
			for (ll=0;ll<(2*matsize+1);ll++)
				s_fishermatrix[kk*(2*matsize+1)+ll]+=s_temp[kk]*s_temp[ll]/model;
	}

	for (kk=0;kk<MATMEM;kk++){

		s_temp[kk]=0;
	}
	/*-----------------------------------------------------------------------------------------------------------------------
	-----------------------------------------------------------------------------------------------------------------------*/
	//copy fisher information matrix into device memory for output.

	//modification of fisher information matrix
	matsz=matsize*2+1;
	if(matsize>0)
	{
		//copy fisher matrix
		for (ii=0;ii<matsz;ii++)
			for(kk=0;kk<matsz;kk++)
				zerofisher[kk*(2*matsize+1)+ii]=s_fishermatrix[kk*(2*matsize+1)+ii];

		//set covariance to zero
		for(kk=0;kk<matsize;kk++){
			for(jj=0;jj<matsize;jj++)
			{
				if(kk!=jj)
				{
					zerofisher[(2*kk+1)*(2*matsize+1)+(2*jj+1)]=0;
					zerofisher[(2*kk)*(2*matsize+1)+(2*jj)]=0;
				}
			}
		}

		// ********************************************************
		//inverse 0 fisher
		kernel_MatInvN(zerofisher, s_temp, s_Diag, matsz);

		//Calculate new fisher
		for(kk=0;kk<matsize;kk++)
		{
			s_crlbx[kk]=sqrt(fabs( s_temp[2*kk*matsz+2*kk]));        //crlbx
			s_crlby[kk]=sqrt(fabs( s_temp[(2*kk+1)*matsz+(2*kk+1)])); //crlby
		}


		for(kk=0;kk<matsize;kk++)
		{
			for(jj=0;jj<matsize;jj++)
			{
				if(kk!=jj)
				{
					disx=pow((fabs( (s_xarray[kk]-s_xarray[jj]))),2)/s_crlbx[kk]/s_crlbx[jj];
					disy=pow((fabs( (s_yarray[kk]-s_yarray[jj]))),2)/s_crlby[kk]/s_crlby[jj];

					s_fishermatrix[(2*kk+1)*(2*matsize+1)+(2*jj+1)]=disy/(disy+1)*s_fishermatrix[(2*kk+1)*(2*matsize+1)+(2*jj+1)];
					s_fishermatrix[(2*kk)*(2*matsize+1)+(2*jj)]=disx/(disx+1)*s_fishermatrix[(2*kk)*(2*matsize+1)+(2*jj)];

				}
			}

		}

		//inverse new fisher
		// matsize will be a indicator saying how big the matrix is for each thread. matsize*2+1 will be the scale of the matrix.
		// start calculating matrix inverse
		/*-----------------------------------------------------------------------------------------------------------------------
		-----------------------------------------------------------------------------------------------------------------------*/
		for (kk=0;kk<MATMEM;kk++){
			s_temp[kk]=0;
		}

		kernel_MatInvN(s_fishermatrix, s_temp, s_Diag, matsz);

	}
	// finished    
	// copy back to device memory
	numco=0;
	for (kk=0;kk<matsz;kk++)
	{
		d_CRLBarray[(2*fitnum+1)*bx*BlockSize+tx*(2*fitnum+1)+kk]=sqrt(fabs(s_temp[kk+kk*matsz]));
		if ((kk+1)%2==0)
		{
			d_covariance[fitnum*bx*BlockSize+tx*fitnum+numco]=s_temp[kk+(kk-1)*matsz];
			numco++;
		}
	}
	return;
}

