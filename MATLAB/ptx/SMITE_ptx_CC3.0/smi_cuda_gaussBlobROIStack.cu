
#include <hip/hip_runtime.h>
#define pi 3.141592
#define min(a,b)            (((a) < (b)) ? (a) : (b))
#define max(a,b)            (((a) > (b)) ? (a) : (b))

//kernel_guassiansampleblobs(sz,Nframes,d_xarray,d_yarray,d_Narray,d_xsigma,d_ysigma,d_covariance,d_im);  

__global__ void kernel_guassiansampleblobs( const int sz, const int Nframes, const float *d_xarray, const float *d_yarray, const float *d_Narray, const float *d_Barray, const float *d_xsigma, const float *d_ysigma, const float *d_covariance,      float *d_im  ) {
	int tx = threadIdx.x; 
	int bx = blockIdx.x;
    int BlockSize = blockDim.x;
	float x,y,xsigma,ysigma,covariance,N,Bg,A;
	int ii,jj,pixelx,pixely,Idx,Idxii;
	
    //Prevent read/write past end of array
    if ((bx*BlockSize+tx)>=Nframes) return;
    
	//import datas from device to shared memory
    x=d_xarray[bx*BlockSize+tx];
	y=d_yarray[bx*BlockSize+tx];
	N=d_Narray[bx*BlockSize+tx];
    Bg=d_Barray[bx*BlockSize+tx];
	xsigma=d_xsigma[bx*BlockSize+tx];
	ysigma=d_ysigma[bx*BlockSize+tx];
	covariance=d_covariance[bx*BlockSize+tx];
    
    // precalculate for speed
    A = N/(2*pi*xsigma*ysigma*sqrt(1-pow(covariance,2)));
    Idx = bx*BlockSize*sz*sz+tx*sz*sz;
	
    for (ii=0;ii<sz;ii++) {
        pixelx=ii; // sample at pixel center
        Idxii = ii*sz;
        for(jj=0;jj<sz;jj++) {
            pixely=jj; // sample at pixel center
            // generate model for pixel ii jj
            d_im[Idx+Idxii+jj] = Bg + (A* exp( -1/(2*(1-pow(covariance,2))) * ( pow(x-pixelx-0.5,2)/pow(xsigma,2) + pow(y-pixely-0.5,2)/pow(ysigma,2) - 2*covariance*(x-pixelx-0.5)*(y-pixely-0.5)/(xsigma*ysigma) ) ) );
        }
    }
	return;
}


//kernel_guassianintegrateblobs(sz,Nframes,d_xarray,d_yarray,d_Narray,d_xsigma,d_ysigma,d_im);

__global__ void kernel_guassianintegrateblobs( const int sz, const int Nframes, const float *d_xarray, const float *d_yarray, const float *d_Narray, const float *d_Barray, const float *d_xsigma, const float *d_ysigma,      float *d_im  ) {

	int tx = threadIdx.x; //matrix number index
	int bx = blockIdx.x;
    int BlockSize = blockDim.x;
	float x,y,xsigma,ysigma,N,Bg,A,B,SX,SY;
	int ii,jj,pixelx,pixely,Idx,Idxii;

    //Prevent read/write past end of array
    if ((bx*BlockSize+tx)>=Nframes) return;

	//import datas from device to shared memory
	x=d_xarray[bx*BlockSize+tx];
	y=d_yarray[bx*BlockSize+tx];
	N=d_Narray[bx*BlockSize+tx];
    Bg=d_Barray[bx*BlockSize+tx];
	xsigma=d_xsigma[bx*BlockSize+tx];
	ysigma=d_ysigma[bx*BlockSize+tx];

    // precalculate these for speed
    A = N/4;
    Idx = bx*BlockSize*sz*sz+tx*sz*sz;
    SX = 1/sqrt(2*pow(xsigma,2));
    SY = 1/sqrt(2*pow(ysigma,2));
    
	for (ii=0;ii<sz;ii++) {
        pixelx=ii;
        Idxii = ii*sz;
        B = erf((x-pixelx)*SX)-erf((x-pixelx-1)*SX);
        for(jj=0;jj<sz;jj++) {
            pixely=jj;
            // generate model for pixel ii jj
            d_im[Idx+Idxii+jj] = Bg + (A*B*(erf((y-pixely)*SY)-erf((y-pixely-1)*SY)));  
        }
    }
	return;
}



