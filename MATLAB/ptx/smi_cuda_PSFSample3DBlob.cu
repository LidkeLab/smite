
#include <hip/hip_runtime.h>

#include "math.h"

__global__ void cuda_PSFSample3DBlob(const double *PX_in, const double *PY_in, const double *PZ_in, const double *PIntensity_in, const double *PBackground_in, const double XYSamPerPix, const double ZSamPerUnit, const int Row, const int Cent, const int *NN, const double *NotPSF, const int NFrames, float *M)
{

	// This function makes the model. It gets the position of the particle and then returns the image. This function is being called inside the loop, several times per iteration.
	// PX, PY, PZ, PIntensity, PBackground are respectively the X-position, Y-position, Z-position and intensity of the particle. PBackground is the background noise.
	// XYSamPerPix is the number of samples (samples of PFS) per pixel along the X and Y axes. ZSamPerUnit is the number of samples (PSF samples) along the Z-axis.
	// Row is the number of pixels along the X and Y axes, assuming that the image is a square image. Cent is the idex of the central column and central row of the NotPSF supposing that we have a square image.
	// NN is a 3-vector which has the 3 dimensions of the NotPSf array. The first element is the number of rows, the second element is the number of columns and the last one is the number of planes along the Z-axis.
	// NotPSF is a 3D array that contains the result of integrating the PSF samples. M is the output of the function, which is the model of the image. 

	double PX, PY, PZ, PIntensity, PBackground;
	double IntX, IntY, DecX, DecY, Axx, Ayy, XYL, YLowInd, XLowInd, PIndX1Y1, PIndX1Y2, PIndX2Y1, PIndX2Y2, CentZ, CeilZ;
	double IntZ, DecZ, Az, DecDecZ, DecDecX, DecDecY, PInterpolX, PInterpolY, PInterpolZ;
	int IndX1Y1, IndX1Y2, IndX2Y1, IndX2Y2, IndZ1CentZ, IndZ2CentZ, IndZ1, IndZ2;
	int Ind1, Ind2, Ind3, Ind4, Ind5, Ind6, Ind7, Ind8;
	int PixNum = Row*Row;

	// The three components of NN are given to the three following parameters.
	int NS1 = (int)NN[0];
	int NS2 = (int)NN[1];
	int NS3 = (int)NN[2];

	// Nn gives the number of pixels inside one of the planes along the Z-axis in the NotPSF variable.
	int Nn = NS1*NS2;

	int ThreadNumPerBlock = blockDim.x;
	int ThreadNumInBlock = threadIdx.x;
	int BlockNum = blockIdx.x;
	int Start = (BlockNum*ThreadNumPerBlock + ThreadNumInBlock)*PixNum;
	int PStart = BlockNum*ThreadNumPerBlock + ThreadNumInBlock;

    if (Start>((NFrames*PixNum)-1)) return;
        
	PX = PX_in[PStart]+1.0f/(2.0f*XYSamPerPix);
	PY = PY_in[PStart]+1.0f/(2.0f*XYSamPerPix);
	PZ = PZ_in[PStart];
	PIntensity = PIntensity_in[PStart];
	PBackground = PBackground_in[PStart];

	
    IntZ = floor(PZ);
	DecZ = PZ - IntZ;
	Az = floor(ZSamPerUnit*DecZ);
	DecDecZ = DecZ - Az / ZSamPerUnit;
	// I used the following three lines rather using round() function.
	CentZ = NS3 / 2.0f;
	CeilZ = ceil(CentZ);
	CentZ = CeilZ;
	// IndZ1 and IndZ2 are the indices of the first components of the PsfZ1 and PsfZ2.
	IndZ1 = (int)round(ZSamPerUnit*(PZ - DecDecZ));
	IndZ2 = IndZ1 + 1;
	IndZ1CentZ = IndZ1 + (int)CentZ;
	IndZ2CentZ = IndZ2 + (int)CentZ;

	IntX = floor(PX);
	IntY = floor(PY);
	DecX = PX - IntX;
	DecY = PY - IntY;
	Axx = floor(XYSamPerPix*DecX);
	Ayy = floor(XYSamPerPix*DecY);
	DecDecX = DecX - Axx / XYSamPerPix;
	DecDecY = DecY - Ayy / XYSamPerPix;

	//XLowInd and YLowInd are the indices of the first components the we pick from PsfZ along the X and Y axes.
	XYL = ceil(XYSamPerPix / 2);
	YLowInd = Cent + XYL - XYSamPerPix*(PY - DecDecY);
	XLowInd = Cent + XYL - XYSamPerPix*(PX - DecDecX);

	// The following for parameters are the linear indices of the first componenets for the PsfX1Y1, PsfX1Y2, PsfX2Y1, PsfX2Y2.
	PIndX1Y1 = (XLowInd)*(double)NS1 + YLowInd;
	PIndX1Y2 = (XLowInd)*(double)NS1 + YLowInd - 1.0f;
	PIndX2Y1 = (XLowInd - 1.0f)*(double)NS1 + YLowInd;
	PIndX2Y2 = (XLowInd - 1.0f)*(double)NS1 + YLowInd - 1.0f;

    PInterpolX = XYSamPerPix*(1.0f / XYSamPerPix - DecDecX);
    PInterpolY = XYSamPerPix*(1.0f / XYSamPerPix - DecDecY);
    PInterpolZ = ZSamPerUnit*(1.0f / ZSamPerUnit - DecDecZ);

for (int nn = 0; nn < PixNum; nn++)
	{
		// the interpolation along X and Y axes and gives the final model.
		double w = nn / Row;
		int xx = (int)floor(w);
		int yy = nn - Row*xx;

		IndX1Y1 = (int)PIndX1Y1 + (int)round(XYSamPerPix*(double)(NS1*xx)) + (int)round(XYSamPerPix*(double)yy);
		IndX1Y2 = (int)PIndX1Y2 + (int)round(XYSamPerPix*(double)(NS1*xx)) + (int)round(XYSamPerPix*(double)yy);
		IndX2Y1 = (int)PIndX2Y1 + (int)round(XYSamPerPix*(double)(NS1*xx)) + (int)round(XYSamPerPix*(double)yy);
		IndX2Y2 = (int)PIndX2Y2 + (int)round(XYSamPerPix*(double)(NS1*xx)) + (int)round(XYSamPerPix*(double)yy);
		Ind1 = Nn*(IndZ1CentZ-1)+IndX1Y1;
        Ind2 = Nn*(IndZ2CentZ-1)+IndX1Y1;
        Ind3 = Nn*(IndZ1CentZ-1)+IndX2Y1;
        Ind4 = Nn*(IndZ2CentZ-1)+IndX2Y1;
        Ind5 = Nn*(IndZ1CentZ-1)+IndX1Y2;
        Ind6 = Nn*(IndZ2CentZ-1)+IndX1Y2;
        Ind7 = Nn*(IndZ1CentZ-1)+IndX2Y2;
        Ind8 = Nn*(IndZ2CentZ-1)+IndX2Y2;

		M[Start + nn] = PIntensity*(PInterpolY * (PInterpolX*(PInterpolZ*NotPSF[Ind1] + (1.0f-PInterpolZ)*NotPSF[Ind2])
			+ (1.0f-PInterpolX)*(PInterpolZ*NotPSF[Ind3] + (1.0f-PInterpolZ)*NotPSF[Ind4])) 
            + (1.0f-PInterpolY)*(PInterpolX*(PInterpolZ*NotPSF[Ind5] + (1.0f-PInterpolZ)*NotPSF[Ind6])
			+ (1.0f-PInterpolX)*(PInterpolZ*NotPSF[Ind7] + (1.0f-PInterpolZ)*NotPSF[Ind8]))) + PBackground;

	}
}
