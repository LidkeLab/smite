#include "hip/hip_runtime.h"

//#include "hip/hip_runtime.h"
#include "definitions.h"
//#include "kernel.h"

//#include <stdio.h>
//#include <stdlib.h>
//#include <string.h>

        
        
__global__ void kernel_gaussMajor(float * d, int MajorSize, float b0, float b1, float b2, float b3, float B)
{
	//Gaussian filter along the Major dimension.  
	int MinorSize = blockDim.x;
	int idx = threadIdx.x;
	int idz = blockIdx.x;

	float w0, w1, w2, w3;
	float temp;
	int ii = 0;
	const int base = idz*MinorSize*MajorSize + idx*MajorSize;

	//forward
	w1 = w2 = w3 = d[base];
	for (ii = 0; ii<MajorSize; ii++)
	{
		w0 = d[base + ii];
		temp = w0*B + (b1*w1 + b2*w2 + b3*w3) / b0;
		d[base + ii] = temp;
		w3 = w2;
		w2 = w1;
		w1 = temp;
	}

	//backward
	w1 = w2 = w3 = d[base + MajorSize - 1];
	for (ii = MajorSize - 1; ii >= 0; ii--)
	{
		w0 = d[base + ii];
		temp = w0*B + (b1*w1 + b2*w2 + b3*w3) / b0;
		d[base + ii] = temp;
		w3 = w2;
		w2 = w1;
		w1 = temp;
	}
}


__device__ float kernel_norm(int x, float s1)
{
    return 1/sqrt(2*pi)/s1*exp( - x*x/(2*s1*s1));
}

__global__ void kernel_gaussMajor_sCMOS(const float * d, const float * v, float * d_out, int MajorSize, float Sigma)
{
	//Gaussian filter along the Major dimension.  
    // Noise-Weighted convolution by Gaussians, seperable
    // *d is the data
    // *v is a variance image
    // Sigma is the small kernel
         
    int MinorSize = blockDim.x;
	int idx = threadIdx.x;
	int idz = blockIdx.x;
    
    int st,en; 
	float weight, var, weightsum, varsum;
	int ii = 0;
	const int base = idz*MinorSize*MajorSize + idx*MajorSize;
    const int basev = idx*MajorSize;
    float winsize = 3* Sigma;

    // Each thread does one row.  
    // variance weighted  x =         
    for (ii = 0; ii< MajorSize; ii++){
        st = max(0,floor(ii-winsize));
        en = min(MajorSize-1,ii+winsize) ;
        varsum = 0;
        weightsum = 0;
        for (int jj = st; jj < en; jj++){
            var=kernel_norm(ii-jj,Sigma)/v[basev+jj];
            weight=var*d[base+jj];
            varsum += var;
            weightsum += weight;      
        }
        d_out[base+ii]=weightsum/varsum;
    }
}

__global__ void kernel_gaussMinor(float * d, int MinorSize, float b0, float b1, float b2, float b3, float B)
{
	//this kernel does gaussian filter along the Minor dimension.  
    //Note: faster to permute and use kernel_gaussMajor
	int MajorSize = blockDim.x;
	int idy = threadIdx.x;
	int idz = blockIdx.x;

	float w0, w1, w2, w3;
	float temp;
	int ii = 0;
	const int base = idz*MinorSize*MajorSize + idy;

	//forward
	w1 = w2 = w3 = d[base];
	for (ii = 0; ii<MinorSize; ii++)
	{
		w0 = d[base + ii*MajorSize];
		temp = w0*B + (b1*w1 + b2*w2 + b3*w3) / b0;
		d[base + ii*MajorSize] = temp;
		w3 = w2;
		w2 = w1;
		w1 = temp;
	}

	//backward
	w1 = w2 = w3 = d[base + MajorSize*(MinorSize - 1)];
	for (ii = MinorSize - 1; ii >= 0; ii--)
	{
		w0 = d[base + ii*MajorSize];
		temp = w0*B + (b1*w1 + b2*w2 + b3*w3) / b0;
		d[base + ii*MajorSize] = temp;
		w3 = w2;
		w2 = w1;
		w1 = temp;
	}
}

__global__ void kernel_subtract(float * d_A, float * d_B)
{
	//Subtract second array from first array
    //Note: faster to subtract gpuArrays in MATLAB       
	int Xsize = blockDim.x;
	int Ysize = gridDim.x;
	int idx = threadIdx.x + Xsize*blockIdx.x + Xsize*Ysize*blockIdx.y;
	d_A[idx] = d_A[idx] - d_B[idx];
}

__global__ void kernel_LocalMaxFirstPass(const float * d_A, float * d_B, const int kernelsz, const float minval)
{
	//this kernel does max finding along the Major dimension.  
            //y-major, x-minor
	int MajorSize = blockDim.x;
	int MinorSize = gridDim.x;
	
    int x = blockIdx.x;  //minor
	int y = threadIdx.x; //major
	int z = blockIdx.y;

	//this is the pixel that we are searching around
	int idx = y + MajorSize*x + MinorSize*MajorSize*z;
	
	//define search only up to edges   
    int start = fmaxf(0, y - kernelsz);
	int end = fminf(MajorSize - 1, y + kernelsz);

	float maxval = minval;
	float inpixel = d_A[idx];

	for (int ii = start; ii<=end; ii++) 
		maxval = fmaxf(maxval, d_A[ii+ MajorSize*x + MinorSize*MajorSize*z]);

	//if any other pixel is larger set pixel idx to negative of that value, otherwise keep
    if (maxval>inpixel) d_B[idx]=-maxval;else d_B[idx]=maxval;

}

__global__ void kernel_LocalMaxSecondPass(const float * d_A, float * d_B, const int kernelsz, const float minval)
{
	//Max finding in second dimension, but input array should be permuted
    //so that operation is done along major axis.         
	int MajorSize = blockDim.x; 
	int MinorSize = gridDim.x;

	int x = blockIdx.x;  //minor
	int y = threadIdx.x; //major
	int z = blockIdx.y;

	//this is the pixel that we are searching around
	int idx = y + MajorSize*x + MinorSize*MajorSize*z;

	//define search only up to edges   
    int start = fmaxf(0, y - kernelsz);
	int end = fminf(MajorSize - 1, y + kernelsz);

	float maxval = minval;
	float inpixel = d_A[idx];

	//find the maximum absolute value in the filter window
	for (int ii = start; ii<=end; ii++)
		maxval = fmaxf(maxval, fabsf(d_A[ii+ MajorSize*x + MinorSize*MajorSize*z]));

	//if our pixel under test is equal to maximum, then flag that with '1', otherwise '0'
    if (fabsf(maxval-inpixel)<1e-6)d_B[idx] = (float)1;else d_B[idx] = (float)0;
	
}

